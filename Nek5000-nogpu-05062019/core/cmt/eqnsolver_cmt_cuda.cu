#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <sys/time.h>
#include "nvml.h"
#include "cuda_helpers.h"
//#include "cuda_helpers.h"

__global__ void igtu_cmt_gpu_kernel1(double *flux, int nfq, int toteq,int toteqlxz2ldimlelt,int lxz2ldimlelt,int iuj,double *graduf,double *area,double *unx, double *uny, double *unz,int if3d,int ldim){

	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nfq){

		// int ix = id % lx1;
		// int iy = (id/lx1)%ly1;
		// int iz = (id / (lx1*ly1))%lz1;
		// int e = id/nxyz;


		for(int eq=0;eq<toteq;eq++){

			graduf[2*toteqlxz2ldimlelt+eq*lxz2ldimlelt+id]= flux[(iuj-1)+eq*lxz2ldimlelt+id]*area[id];
			graduf[0*toteqlxz2ldimlelt+eq*lxz2ldimlelt+id]= graduf[2*toteqlxz2ldimlelt+eq*lxz2ldimlelt+id]*unx[id];
			graduf[1*toteqlxz2ldimlelt+eq*lxz2ldimlelt+id]= graduf[2*toteqlxz2ldimlelt+eq*lxz2ldimlelt+id]*uny[id];


			if(if3d){
				graduf[2*toteqlxz2ldimlelt+eq*lxz2ldimlelt+id]=graduf[2*toteqlxz2ldimlelt+eq*lxz2ldimlelt+id]*unz[id];

			}
		}
	}
}

__global__ void igtu_cmt_gpu_kernel2(double *gradu,double *graduf, int eq, int nfq,int *iface_flux,int toteqlxyzlelt,int toteqlxz2ldimlelt,int lxyzlelt,int nf,int ldim,int toteq,int lxyz,int lxz2ldimlelt){

	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nfq){

		int e = id/nf;

		for(int j=0;j<ldim;j++){
			for(int eq2=0;eq2<toteq;eq2++){
				//add_face2full_cmt(nel,nx,ny,nz,iface,vols,faces)
				//something is wrong with the original functions. gradu and graduf does not consider e. lot of overrides in the for loops vols(i,1,1,ie). check with Dr.Tania. adeesha  . gradu has nelt but not in original.

				int newi = iface_flux[id];
				gradu[j*toteqlxyzlelt+eq2*lxyzlelt+e*lxyz+newi ] =  gradu[j*toteqlxyzlelt+eq2*lxyzlelt+e*lxyz+newi ]  + graduf[j*toteqlxz2ldimlelt+eq2*lxz2ldimlelt+id]; 
			}



		}


	}

}

// igtu_cmt_gpu_kernel3<<<gridSize3, blockSize1>>>(d_diffh,d_gradu, d_vtrans,d_vdiff,d_vx,d_vy,d_vz,d_u,d_viscscr,d_superhugeh, d_ur, d_us,d_ut, d_jacmi,d_rxm1,d_rym1, d_rzm1,d_sxm1, d_sym1,d_szm1,d_txm1, d_tym1,d_tzm1, lx1[0],ly1[0],lz1[0], lxy,nxyz,toteq[0], nnel,lxyzlelt,toteqlxyz, toteqlxyzlelt, irho[0], ilam[0],imu[0],icv[0], iknd[0],inus[0],eq,if3d[0],ldim[0] );



__global__ void igtu_cmt_gpu_kernel3(double *diffh,double *gradu, double *vtrans, double *vdiff, double * vx, double *vy, double *vz, double *u, double *viscscr, double *superhugeh, double *ur, double *us, double *ut, double *jacmi, double *rxm1,double *rym1, double *rzm1, double *sxm1, double *sym1, double *szm1,double *txm1, double *tym1, double *tzm1, int lx1,int ly1,int lz1, int lxy, int nxyz,  int toteq, int nnel, int lxyzlelt, int toteqlxyz, int toteqlxyzlelt, int irho, int ilam, int imu, int icv, int iknd, int inus,int eq,int  if3d,int ldim){

	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nnel){

		int ix = id % lx1;
		int iy = (id/lx1)%ly1;
		int iz = (id / (lxy))%lz1;
		int e = id/nxyz;

		//agradu(diffh,gradu,e,eq) // flux =diffh change later. adeesha
		// call fluxj_ns (flux,du,e,eq)
		if(eq < toteq-1){
			if(eq==1){
				//A21kldUldxk(flux(1,1),gradu,e)
				double dU1x = gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU2x=gradu[0*toteqlxyzlelt+1*lxyzlelt+id ];
				double  dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU3y=gradu[1*toteqlxyzlelt+2*lxyzlelt+id ];
				double  dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU4z=gradu[2*toteqlxyzlelt+3*lxyzlelt+id ];
				double  rho   =vtrans[(irho-1)*lxyzlelt+id];
				double  lambda=vdiff[(ilam-1)*lxyzlelt+id];
				double  mu    =vdiff[(imu-1)*lxyzlelt+id];
				double  u1    =vx[id];
				double  u2    =vy[id];
				double  u3    =vz[id];
				double  lambdamu=lambda+2.0*mu;
				diffh[0*lxyzlelt+id]=(lambda*(dU4z+dU3y-u3*dU1z-u2*dU1y)+lambdamu*(dU2x-u1*dU1x))/rho;

				//A22kldUldxk(flux(1,2),gradu,e)
				dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double   dU3x=gradu[0*toteqlxyzlelt+2*lxyzlelt+id ];
				dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double   dU2y=gradu[1*toteqlxyzlelt+1*lxyzlelt+id ];
				rho   =vtrans[(irho-1)*lxyzlelt+id];
				mu    =vdiff[(imu-1)*lxyzlelt+id];
				u1    =vx[id];
				u2    =vy[id];
				diffh[1*lxyzlelt+id]=mu*(dU3x+dU2y-u1*dU1y-u2*dU1x)/rho;

				//A23kldUldxk(flux(1,3),gradu,e)

				dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU4x=gradu[0*toteqlxyzlelt+3*lxyzlelt+id ];
				dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU2z=gradu[2*toteqlxyzlelt+1*lxyzlelt+id ];
				rho   =vtrans[(irho-1)*lxyzlelt+id];
				mu    =vdiff[(imu-1)*lxyzlelt+id];
				u1    =vx[id];
				u3    =vz[id];
				diffh[2*lxyzlelt+id]=mu*(dU4x+dU2z-u1*dU1z-u3*dU1x)/rho;

			}
			else if(eq==2){
				//A31kldUldxk(flux(1,1),gradu,e)
				double dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU3x=gradu[0*toteqlxyzlelt+2*lxyzlelt+id ];
				double dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU2y=gradu[1*toteqlxyzlelt+1*lxyzlelt+id ];
				double rho   =vtrans[(irho-1)*lxyzlelt+id];
				double mu    =vdiff[(imu-1)*lxyzlelt+id];
				double u1    =vx[id];
				double u2    =vy[id];
				diffh[0*lxyzlelt+id] = mu*(dU3x+dU2y-u1*dU1y-u2*dU1x)/rho;

				//A32kldUldxk(flux(1,2),gradu,e)
				dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU2x=gradu[0*toteqlxyzlelt+1*lxyzlelt+id ];
				dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU3y=gradu[1*toteqlxyzlelt+2*lxyzlelt+id ];
				double dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double   dU4z=gradu[2*toteqlxyzlelt+3*lxyzlelt+id ];
				rho   =vtrans[(irho-1)*lxyzlelt+id];;
				double lambda=vdiff[(ilam-1)*lxyzlelt+id];
				mu    =vdiff[(imu-1)*lxyzlelt+id];
				u1    =vx[id];
				u2    =vy[id];
				double u3    =vz[id];
				double lambdamu=lambda+2.0*mu;
				diffh[1*lxyzlelt+id]=(lambda*(dU4z+dU2x-u3*dU1z-u1*dU1x)+lambdamu*(dU3y-u2*dU1y))/rho;

				//A33kldUldxk(flux(1,3),gradu,e)
				dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU4y=gradu[1*toteqlxyzlelt+3*lxyzlelt+id ];
				dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double   dU3z=gradu[2*toteqlxyzlelt+2*lxyzlelt+id ];
				rho   =vtrans[(irho-1)*lxyzlelt+id];
				mu    =vdiff[(imu-1)*lxyzlelt+id];
				u2    =vy[id];
				u3    =vz[id];
				diffh[2*lxyzlelt+id]=mu*(dU4y+dU3z-u2*dU1z-u3*dU1y)/rho;	



			}
			else if(eq==3){
				//A41kldUldxk(flux(1,1),gradu,e)
				double dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU4x=gradu[0*toteqlxyzlelt+3*lxyzlelt+id ];
				double dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU2z=gradu[2*toteqlxyzlelt+1*lxyzlelt+id ];
				double rho   =vtrans[(irho-1)*lxyzlelt+id];
				double mu    =vdiff[(imu-1)*lxyzlelt+id];

				double u1    =vx[id];
				double u3    =vz[id];
				diffh[0*lxyzlelt+id]=mu*(dU4x+dU2z-u1*dU1z-u3*dU1x)/rho;

				//A42kldUldxk(flux(1,2),gradu,e)
				double dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU4y=gradu[1*toteqlxyzlelt+3*lxyzlelt+id ];
				dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU3z=gradu[2*toteqlxyzlelt+2*lxyzlelt+id ];
				rho   =vtrans[(irho-1)*lxyzlelt+id];
				mu    =vdiff[(imu-1)*lxyzlelt+id];

				double u2    =vy[id];
				u3    =vz[id];
				diffh[1*lxyzlelt+id]=mu*(dU4y+dU3z-u2*dU1z-u3*dU1y)/rho;

				//A43kldUldxk(flux(1,3),gradu,e)
				dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU2x=gradu[0*toteqlxyzlelt+1*lxyzlelt+id ];
				dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU3y=gradu[1*toteqlxyzlelt+2*lxyzlelt+id ];
				dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU4z=gradu[2*toteqlxyzlelt+3*lxyzlelt+id ];
				rho   =vtrans[(irho-1)*lxyzlelt+id];
				double lambda=vdiff[(ilam-1)*lxyzlelt+id];
				mu    =vdiff[(imu-1)*lxyzlelt+id];
				u1    =vx[id];
				u2    =vy[id];
				u3    =vz[id];
				double lambdamu=lambda+2.0*mu;
				diffh[2*lxyzlelt+id]=(lambda*(dU3y+dU2x-u2*dU1y-u1*dU1x)+lambdamu*(dU4z-u3*dU1z))/rho;	
			}

		}
		else{
			if(if3d){
				//a53kldUldxk(flux(1,3),gradu,e)
				double dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU2x=gradu[0*toteqlxyzlelt+1*lxyzlelt+id ];
				double  dU3x=gradu[0*toteqlxyzlelt+2*lxyzlelt+id ];
				double  dU4x=gradu[0*toteqlxyzlelt+3*lxyzlelt+id ];
				double  dU5x=gradu[0*toteqlxyzlelt+4*lxyzlelt+id ];
				double  dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU2y=gradu[1*toteqlxyzlelt+1*lxyzlelt+id ];
				double  dU3y=gradu[1*toteqlxyzlelt+2*lxyzlelt+id ];
				double  dU4y=gradu[1*toteqlxyzlelt+3*lxyzlelt+id ];
				double  dU5y=gradu[1*toteqlxyzlelt+4*lxyzlelt+id ];
				double dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU2z=gradu[2*toteqlxyzlelt+1*lxyzlelt+id ];
				double  dU3z=gradu[2*toteqlxyzlelt+2*lxyzlelt+id ];
				double  dU4z=gradu[2*toteqlxyzlelt+3*lxyzlelt+id ];
				double  dU5z=gradu[2*toteqlxyzlelt+4*lxyzlelt+id ];
				double  rho   =vtrans[(irho-1)*lxyzlelt+id];
				double  cv    =vtrans[(icv-1)*lxyzlelt+id]/rho;
				double  lambda=vdiff[(ilam-1)*lxyzlelt+id];
				double  mu    =vdiff[(imu-1)*lxyzlelt+id];
				double K     =vdiff[(iknd-1)*lxyzlelt+id];;
				double  u1    =vx[id];
				double  u2    =vy[id];
				double  u3    =vz[id];
				double  E     =u[e*toteqlxyz+(toteq-1)*nxyz+iz*lxy+iy*lx1+ix]/rho;
				double lambdamu=lambda+mu;
				double kmcvmu=K-cv*mu;
				diffh[2*lxyzlelt+id]=(K*(dU5z-E*dU1z)+cv*u3*(lambda*dU4z+2*mu*dU4z+lambda*dU3y+lambda*dU2x)-K*u3*dU4z+cv*mu*u2*(dU4y+dU3z)+cv*mu*u1*(dU4x+dU2z)-K*u2*dU3z-K*u1*dU2z-cv*(lambda+2*mu)*u3*u3*dU1z+K*u3*u3*dU1z+ K*u2*u2*dU1z-cv*mu*u2*u2*dU1z+K*u1*u1*dU1z-cv*mu*u1*u1*dU1z-cv*(lambda+mu)*u2*u3*dU1y-cv*(lambda+mu)*u1*u3*dU1x)/(cv*rho);

			}
			else{
				for(int kfortoteq=0;kfortoteq<toteq;kfortoteq++){
					gradu[2*toteqlxyzlelt+kfortoteq*lxyzlelt+id]=0;
				}
				vz[id]=0;

			}


		}
		//a51kldUldxk(flux(1,1),gradu,e)
		double dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
		double dU2x=gradu[0*toteqlxyzlelt+1*lxyzlelt+id ];
		double  dU3x=gradu[0*toteqlxyzlelt+2*lxyzlelt+id ];
		double  dU4x=gradu[0*toteqlxyzlelt+3*lxyzlelt+id ];
		double  dU5x=gradu[0*toteqlxyzlelt+4*lxyzlelt+id ];
		double  dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
		double  dU2y=gradu[1*toteqlxyzlelt+1*lxyzlelt+id ];
		double  dU3y=gradu[1*toteqlxyzlelt+2*lxyzlelt+id ];
		double  dU4y=gradu[1*toteqlxyzlelt+3*lxyzlelt+id ];
		double  dU5y=gradu[1*toteq*toteqlxyzlelt+4*lxyzlelt+id ];
		double dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
		double  dU2z=gradu[2*toteqlxyzlelt+1*lxyzlelt+id ];
		double  dU3z=gradu[2*toteqlxyzlelt+2*lxyzlelt+id ];
		double  dU4z=gradu[2*toteqlxyzlelt+3*lxyzlelt+id ];
		double  dU5z=gradu[2*toteqlxyzlelt+4*lxyzlelt+id ];
		double  rho   =vtrans[(irho-1)*lxyzlelt+id];
		double  cv    =vtrans[(icv-1)*lxyzlelt+id]/rho;
		double  lambda=vdiff[(ilam-1)*lxyzlelt+id];
		double  mu    =vdiff[(imu-1)*lxyzlelt+id];
		double K     =vdiff[(iknd-1)*lxyzlelt+id];;
		double  u1    =vx[id];
		double  u2    =vy[id];
		double  u3    =vz[id];
		double  E     =u[e*toteqlxyz+(toteq-1)*nxyz+iz*lxy+iy*lx1+ix]/rho;
		double lambdamu=lambda+mu;
		double kmcvmu=K-cv*mu;
		diffh[0*lxyzlelt+id]=(K*dU5x+cv*lambda*u1*dU4z-kmcvmu*u3*dU4x+cv*lambda*u1*dU3y-kmcvmu*u2*dU3x+cv*mu*u3*dU2z+cv*mu*u2*dU2y+(cv*lambda-K+2*cv*mu)*u1*dU2x-cv*lambdamu*u1*u3*dU1z-cv*lambdamu*u1*u2*dU1y+(K*u3*u3-cv*mu*u3*u3+K*u2*u2-cv*mu*u2*u2-cv*lambda*u1*u1+K*u1*u1-2*cv*mu*u1*u1-E*K)*dU1x)/(cv*rho);

		//a52kldUldxk(flux(1,2),gradu,e)
		dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
		dU2x=gradu[0*toteqlxyzlelt+1*lxyzlelt+id ];
		dU3x=gradu[0*toteqlxyzlelt+2*lxyzlelt+id ];
		dU4x=gradu[0*toteqlxyzlelt+3*lxyzlelt+id ];
		dU5x=gradu[0*toteqlxyzlelt+4*lxyzlelt+id ];
		dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
		dU2y=gradu[1*toteqlxyzlelt+1*lxyzlelt+id ];
		dU3y=gradu[1*toteqlxyzlelt+2*lxyzlelt+id ];
		dU4y=gradu[1*toteqlxyzlelt+3*lxyzlelt+id ];
		dU5y=gradu[1*toteqlxyzlelt+4*lxyzlelt+id ];
		dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
		dU2z=gradu[2*toteqlxyzlelt+1*lxyzlelt+id ];
		dU3z=gradu[2*toteqlxyzlelt+2*lxyzlelt+id ];
		dU4z=gradu[2*toteqlxyzlelt+3*lxyzlelt+id ];
		dU5z=gradu[2*toteqlxyzlelt+4*lxyzlelt+id ];
		rho   =vtrans[(irho-1)*lxyzlelt+id];
		cv    =vtrans[(icv-1)*lxyzlelt+id]/rho;
		lambda=vdiff[(ilam-1)*lxyzlelt+id];
		mu    =vdiff[(imu-1)*lxyzlelt+id];
		K     =vdiff[(iknd-1)*lxyzlelt+id];;
		u1    =vx[id];
		u2    =vy[id];
		u3    =vz[id];
		E     =u[e*toteqlxyz+(toteq-1)*nxyz+iz*lxy+iy*lx1+ix]/rho;
		lambdamu=lambda+mu;
		kmcvmu=K-cv*mu;
		diffh[1*lxyzlelt+id]=(K*dU5y+cv*lambda*u2*dU4z-kmcvmu*u3*dU4y+cv*mu*u3*dU3z+(cv*lambda-K+2*cv*mu)*u2*dU3y+cv*mu*u1*dU3x-kmcvmu*u1*dU2y+cv*lambda*u2*dU2x-cv*lambdamu*u2*u3*dU1z+(K*u3*u3-cv*mu*u3*u3-cv*lambda*u2*u2+K*u2*u2-2*cv*mu*u2*u2+K*u1*u1-cv*mu*u1*u1-E*K)*dU1y-cv*lambdamu*u1*u2*dU1x)/(cv*rho);


		//call fluxj_evm(flux,du,e,eq)

		if(eq==0){
			for(int jj=0;jj<ldim;jj++){
				diffh[jj*lxyzlelt+id]=  diffh[jj*lxyzlelt+id]+vdiff[(inus-1)*lxyzlelt+id]*gradu[jj*toteqlxyzlelt+id];
			}

		}
		else{
			if(eq<toteq-1){
				viscscr[id]=gradu[0*toteqlxyzlelt+(eq-1)*lxyzlelt+id ]; // problem with du indices. du(1,1,eq-1) third is for ldim check wih Dr.Tania adeesha.
				viscscr[id]=viscscr[id]*vdiff[(inus-1)*lxyzlelt+id];
				diffh[0*lxyzlelt+id]=  diffh[0*lxyzlelt+id]+viscscr[id]*vx[id]; 
				diffh[1*lxyzlelt+id]=  diffh[1*lxyzlelt+id]+viscscr[id]*vy[id];
				if(if3d){
					diffh[2*lxyzlelt+id]=  diffh[2*lxyzlelt+id]+viscscr[id]*vz[id];
				} 

			}
			else{
				if(if3d){
					viscscr[id] = vx[id]*vx[id]+vy[id]*vy[id]+vz[id]*vz[id];
				}
				else{
					viscscr[id] = vx[id]*vx[id]+vy[id]*vy[id];
				}
				viscscr[id] =  viscscr[id] *vdiff[(inus-1)*lxyzlelt+id];
				for(int jj=0;jj<ldim;jj++){
					diffh[jj*lxyzlelt+id]=  diffh[jj*lxyzlelt+id]+viscscr[id] *gradu[jj*toteqlxyzlelt+id];
				}
				for(int jj=0;jj<ldim;jj++){
					for(int eq2=1;eq2<ldim+1;eq2++){
						viscscr[id]=gradu[jj*toteqlxyzlelt+eq2*lxyzlelt+id ]* u[e*toteqlxyz+eq2*nxyz+iz*lxy+iy*lx1+ix]+vdiff[(inus-1)*lxyzlelt+id] ;
						viscscr[id]=viscscr[id]/vtrans[(irho-1)*lxyzlelt+id];	
						diffh[jj*lxyzlelt+id]=diffh[jj*lxyzlelt+id]-viscscr[id];
						diffh[jj*lxyzlelt+id]=  diffh[jj*lxyzlelt+id]+vdiff[(inus-1)*lxyzlelt+id] *gradu[jj*toteqlxyzlelt+(toteq-1)*lxyzlelt+id ];
					}

				} 



			}
		}
		//end of agradu
		for(int j=0;j<ldim;j++){
			superhugeh[j*lxyzlelt+id] = diffh[j*lxyzlelt+id];
		}

		if(if3d){
			ur[id] =  jacmi[id] *( rxm1[id]*superhugeh[0*lxyzlelt+id]+ rym1[id]*superhugeh[1*lxyzlelt+id]+rzm1[id]*superhugeh[2*lxyzlelt+id]);
			us[id] =  jacmi[id] *( sxm1[id]*superhugeh[0*lxyzlelt+id]+ sym1[id]*superhugeh[1*lxyzlelt+id]+szm1[id]*superhugeh[2*lxyzlelt+id]);
			ut[id] =  jacmi[id] *( txm1[id]*superhugeh[0*lxyzlelt+id]+ tym1[id]*superhugeh[1*lxyzlelt+id]+tzm1[id]*superhugeh[2*lxyzlelt+id]);

		}
		else{
			ur[id] =  jacmi[id] *( rxm1[id]*superhugeh[0*lxyzlelt+id]+ rym1[id]*superhugeh[1*lxyzlelt+id]);
			us[id] =  jacmi[id] *( sxm1[id]*superhugeh[0*lxyzlelt+id]+sym1[id]*superhugeh[1*lxyzlelt+id]);
		}





	}

}

__global__ void igtu_cmt_gpu_kernel4(double *res1, double *gradm1_t_overwrites, int nnel, int lx1, int ly1, int lz1,int lxy, int nxyz, double consta, int eq, int lxyzlelt){

	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nnel){

		int ix = id % lx1;
		int iy = (id/lx1)%ly1;
		int iz = (id / (lxy))%lz1;
		int e = id/nxyz;
		gradm1_t_overwrites[id]= gradm1_t_overwrites[id]*consta;
		res1[eq*lxyzlelt+id]  = res1[eq*lxyzlelt+id] +gradm1_t_overwrites[id];
	}
}
/*void gpu_local_grad3_t(double *u, double *ur, double *us, double *ut, int nxd, double *d, double *dt, double *w, int nel){

  int nxd_2 = nxd * nxd;
  int nxd_3 = nxd_2 * nxd;
// u(nxd,nxd*nxd) = dt(nxd,nxd) * ur(nxd, nxd*nxd) fortran
// u(nxd*nxd,nxd) = ur(nxd*nxd, nxd) * dt(nxd,nxd) C
int blockSize=1024, gridSize;
hipStream_t stream;
hipStreamCreate( &stream );
const double alpha = 1;
const double beta = 0;

gridSize = (int)ceil((float)nel*nxd_3/blockSize);
//mxm<<<gridSize, blockSize>>>(ur,nxd_2, dt, nxd, u, nxd, nel, nxd_3, 0, nxd_3, 0);
cuda_multi_gemm_unif(stream, 'N', 'N', nxd, nxd, nxd_2, &alpha, dt, nxd, 0, ur, nxd, nxd_3, &beta, u, nxd, nxd_3, nel, gridSize);

for(int k = 0; k<nxd;k++){
//wk(nxd,nxd) = usk(nxd,nxd)*D(nxd,nxd) fortran
//wk(nxd,nxd) = D(nxd,nxd)*usk(nxd,nxd) C
gridSize = (int)ceil((float)nel*nxd_2/blockSize);
//mxm<<<gridSize, blockSize>>>(d,nxd, us+k*nxd_2, nxd, w+k*nxd_2, nxd, nel, 0, nxd_3, nxd_3, 0);
cuda_multi_gemm_unif(stream, 'N', 'N', nxd, nxd, nxd, &alpha, us+k*nxd_2, nxd, nxd_3, d, nxd, 0, &beta, w+k*nxd_2, nxd, nxd_3, nel, gridSize);


}
gridSize = (int)ceil((float)nel*nxd_3/blockSize);
nekadd2<<<gridSize, blockSize>>>(u,w, nel*nxd_3);
//w(nxd*nxd,nxd) = ut(nxd*nxd,nxd) * D(nxd,nxd) fortran
//w(nxd,nxd*nxd) = D(nxd,nxd) * ut(nxd,nxd*nxd) C
//mxm<<<gridSize, blockSize>>>(d,nxd, ut, nxd, w, nxd_2, nel, 0, nxd_3, nxd_3, 0);
cuda_multi_gemm_unif(stream, 'N', 'N', nxd_2, nxd, nxd, &alpha, ut, nxd, nxd_3, d, nxd, 0, &beta, w, nxd_2, nxd_3, nel, gridSize);

nekadd2<<<gridSize, blockSize>>>(u,w, nel*nxd_3);
hipStreamDestroy(stream);



}

void gpu_local_grad2_t(double *u, double *ur, double *us, double *ut, int nxd, double *d, double *dt, double *w, int nel){

int nxd_2 = nxd * nxd;
int nxd_3 = nxd_2 * nxd;
// u(nxd,nxd*nxd) = dt(nxd,nxd) * ur(nxd, nxd*nxd) fortran
// u(nxd*nxd,nxd) = ur(nxd*nxd, nxd) * dt(nxd,nxd) C
int blockSize=1024, gridSize;
hipStream_t stream;
hipStreamCreate( &stream );
const double alpha = 1;
const double beta = 0;

gridSize = (int)ceil((float)nel*nxd_3/blockSize);
//mxm<<<gridSize, blockSize>>>(ur,nxd_2, dt, nxd, u, nxd, nel, nxd_3, 0, nxd_3, 0);
cuda_multi_gemm_unif(stream, 'N', 'N', nxd, nxd, nxd_2, &alpha, dt, nxd, 0, ur, nxd, nxd_3, &beta, u, nxd, nxd_3, nel, gridSize);

gridSize = (int)ceil((float)nel*nxd_3/blockSize);
//w(nxd*nxd,nxd) = ut(nxd*nxd,nxd) * D(nxd,nxd) fortran
//w(nxd,nxd*nxd) = D(nxd,nxd) * ut(nxd,nxd*nxd) C
//mxm<<<gridSize, blockSize>>>(d,nxd, ut, nxd, w, nxd_2, nel, 0, nxd_3, nxd_3, 0);
cuda_multi_gemm_unif(stream, 'N', 'N', nxd_2, nxd, nxd, &alpha, ut, nxd, nxd_3, d, nxd, 0, &beta, w, nxd_2, nxd_3, nel, gridSize);

nekadd2<<<gridSize, blockSize>>>(u,w, nel*nxd_3);
hipStreamDestroy(stream);


}

 */


extern "C" void igtu_cmt_gpu_wrapper_(int *glbblockSize1,int *glbblockSize2,double *d_flux,double *d_gradu,double *d_graduf, int *d_iface_flux,double *d_diffh,double *d_vtrans,double *d_vdiff,double *d_vx,double *d_vy,double *d_vz,double *d_u,double *d_viscscr, double *d_jacmi,double *d_rxm1,double *d_rym1, double *d_rzm1,double *d_sxm1, double *d_sym1,double *d_szm1,double *d_txm1, double *d_tym1,double *d_tzm1,double *d_dxm1,double *d_dxtm1,double *d_res1,int *toteq,int *iuj,int *lx1,int *ly1,int *lz1,int *irho, int *ilam,int *imu,int *icv, int *iknd,int *inus,int *nelt,int *lelt, int *ldim,int *ifsip,double *d_area,double *d_unx,double *d_uny,double *d_unz,int *if3d){


	int nxz = lx1[0]*lz1[0];
	int nfaces = 2*ldim[0];
	int nf = nxz*nfaces;// ! 1 element's face points
	int nfq = nf*nelt[0];// ! all points in a pile of faces
	int nnel = lx1[0]*ly1[0]*lz1[0]*nelt[0];
	int nlel = lx1[0]*ly1[0]*lz1[0]*lelt[0];
	int lxy=lx1[0]*ly1[0];
	int lxyzlelt=nlel;

	int nxyz  =lx1[0]*ly1[0]*lz1[0];
	int toteqlxyz=toteq[0]*nxyz;
	int nvol  =nxyz*nelt[0];
	int ngradu=nxyz*toteq[0]*3;

	int lxz2ldimlelt=nf*lelt[0];
	int toteqlxz2ldimlelt= toteq[0]*lxz2ldimlelt;
	int toteqlxyzlelt= toteq[0]*nlel;
	double consta;
	if (ifsip[0]){
		consta=-1.0;// ! SIP
	}
	else{
		consta=1.0;// ! Baumann-Oden
	}



	int blockSize1 = glbblockSize1[0], blockSize2= glbblockSize2[0],gridSize1,gridSize2,gridSize3;
	hipDeviceSynchronize();

	hipError_t code1 = hipPeekAtLastError();
	//        if (code1 != hipSuccess){
	printf("CUDA: Start igtu_cmt_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));
	printf("CUDA: Start igtu_cmt_gpu_wrapper values toteq=%d,iuj=%d,lx1=%d,ly1=%d,lz1=%d,irho=%d,ilam=%d,imu=%d,icv=%d,iknd=%d,inus=%d,nelt=%d,lelt=%d,ldim=%d,ifsip=%d,if3d=%d \n",toteq[0],iuj[0],lx1[0],ly1[0],lz1[0],irho[0],ilam[0],imu[0],icv[0],iknd[0],inus[0],nelt[0],lelt[0],ldim[0],ifsip[0],if3d[0]);
	//      }

	gridSize1 = (int)ceil((float)nfq/blockSize2);
	igtu_cmt_gpu_kernel1<<<gridSize1, blockSize2>>>(d_flux, nfq, toteq[0],toteqlxz2ldimlelt,lxz2ldimlelt,iuj[0],d_graduf,d_area,d_unx,d_uny,d_unz,if3d[0],ldim[0]);
	hipDeviceSynchronize();
	code1 = hipPeekAtLastError();
	printf("CUDA: igtu_cmt_gpu_wrapper after kernel1 cuda status: %s\n",hipGetErrorString(code1));

	double *d_superhugeh;
	double *d_gradm1_t_overwrites;
	double *d_ur;
	double *d_us;
	double *d_ut;
	double *d_tmp;

	hipMalloc((void**)&d_gradm1_t_overwrites,nlel*sizeof(double));
	hipMalloc((void**)&d_superhugeh,nlel*3*sizeof(double));
	hipMalloc((void**)&d_us,nlel*sizeof(double));
	hipMalloc((void**)&d_ut,nlel*sizeof(double));
	hipMalloc((void**)&d_ur,nlel*sizeof(double));
	hipMalloc((void**)&d_tmp,nlel*sizeof(double));
	for(int eq=0;eq<toteq[0];eq++){

		hipMemset(d_superhugeh,0.0, nlel*3*sizeof(double));
		if (eq == 3 && !if3d){}
		else{
			hipMemset(d_gradu,0.0, toteq[0]*3*nlel*sizeof(double));
			hipMemset(d_diffh,0.0, nlel*3*sizeof(double));
			hipMemset(d_ur, 0.0, nlel*sizeof(double));
			hipMemset(d_us, 0.0, nlel*sizeof(double));
			hipMemset(d_ut, 0.0, nlel*sizeof(double));
			hipMemset(d_tmp, 0.0, nlel*sizeof(double));




			gridSize2 = (int)ceil((float)nfq/blockSize2);
			igtu_cmt_gpu_kernel2<<<gridSize2, blockSize2>>>(d_gradu,d_graduf,eq, nfq,d_iface_flux,toteqlxyzlelt,toteqlxz2ldimlelt,lxyzlelt,nf,ldim[0],toteq[0],nxyz,lxz2ldimlelt);


			hipDeviceSynchronize();
			code1 = hipPeekAtLastError();
			printf("CUDA: igtu_cmt_gpu_wrapper after kernel2 cuda status: %s\n",hipGetErrorString(code1));


			gridSize3 = (int)ceil((float)nnel/blockSize1);
			igtu_cmt_gpu_kernel3<<<gridSize3, blockSize1>>>(d_diffh,d_gradu, d_vtrans,d_vdiff,d_vx,d_vy,d_vz,d_u,d_viscscr,d_superhugeh, d_ur, d_us,d_ut, d_jacmi,d_rxm1,d_rym1, d_rzm1,d_sxm1, d_sym1,d_szm1,d_txm1, d_tym1,d_tzm1, lx1[0],ly1[0],lz1[0], lxy,nxyz,toteq[0], nnel,lxyzlelt,toteqlxyz, toteqlxyzlelt, irho[0], ilam[0],imu[0],icv[0], iknd[0],inus[0],eq,if3d[0],ldim[0] );

			hipDeviceSynchronize();
			code1 = hipPeekAtLastError();
			printf("CUDA: igtu_cmt_gpu_wrapper after kernel3 cuda status: %s\n",hipGetErrorString(code1));
			//computation of ur us ut are in the kernel3
			//gradm1_t
			if(if3d){
				gpu_local_grad3_t(d_gradm1_t_overwrites, d_ur, d_us,d_ut,lx1[0],d_dxm1,d_dxtm1, d_tmp, nelt[0]);		
			}
			else{
				gpu_local_grad2_t(d_gradm1_t_overwrites, d_ur, d_us,lx1[0],d_dxm1,d_dxtm1, d_tmp, nelt[0]);		

			}
			hipDeviceSynchronize();
			code1 = hipPeekAtLastError();
			printf("CUDA: igtu_cmt_gpu_wrapper after local_grad cuda status: %s\n",hipGetErrorString(code1));

			gridSize3 = (int)ceil((float)nnel/blockSize1);
			igtu_cmt_gpu_kernel4<<<gridSize3, blockSize1>>>(d_res1, d_gradm1_t_overwrites,  nnel, lx1[0], ly1[0],lz1[0],lxy,nxyz, consta, eq, lxyzlelt);

			hipDeviceSynchronize();
			code1 = hipPeekAtLastError();
			printf("CUDA: igtu_cmt_gpu_wrapper after kernel4 cuda status: %s\n",hipGetErrorString(code1));

		}	
	}
	hipFree(d_ur);
	hipFree(d_us);
	hipFree(d_ut);
	hipFree(d_tmp);
	hipFree(d_superhugeh);
	hipFree(d_gradm1_t_overwrites);

	hipError_t code2 = hipPeekAtLastError();
	//if (code2 != hipSuccess){
	printf("End igtu_cmt_gpu_wrapper cuda status: %s\n",hipGetErrorString(code2));
	// }



}
__global__ void cmtusrf_gpu_kernel(double *usrf,double *xm1,double *ym1,double *zm1,double *vx,double *vy,double *vz,double *t,double *pr,double *sii,double *siii,double *vdiff,double *vtrans,char *cb,double *ptw,int *lglel,int *gllel,double *rhs_fluidp,double *u,double *phig,int nnel,int lx1,int ly1,int lz1,int lxy,int nxyz,int lxyzlelt,int toteqlxyz,int  istep,int  npscal,int two_way,int time_delay,int icmtp,int nlel,int p0th,int ifield){

	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nnel){

		int ix = id % lx1;
		int iy = (id/lx1)%ly1;
		int iz = (id / (lxy))%lz1;
		int e = id/nxyz;

		if(istep==0&&e==0){
			usrf[5*(iz*(lxy)+iy*lx1+ix)+0]=0;
			usrf[5*(iz*(lxy)+iy*lx1+ix)+1]=0;
			usrf[5*(iz*(lxy)+iy*lx1+ix)+2]=0;
			usrf[5*(iz*(lxy)+iy*lx1+ix)+3]=0;
			usrf[5*(iz*(lxy)+iy*lx1+ix)+4]=0;
		}

		int eg=lglel[e];
		//nek assign
		double x = xm1[e*nxyz+iz*lxy+iy*lx1+ix];
		double y = ym1[e*nxyz+iz*lxy+iy*lx1+ix];
		double z = zm1[e*nxyz+iz*lxy+iy*lx1+ix];
		double r = x*x+y*y;
		double theta=0.0;
		if (r>0.0){ r = sqrtf(r);}
		if ( x != 0.0 || y!= 0.0){theta = atan2(y,x);   }
		double ux= vx[e*nxyz+iz*lxy+iy*lx1+ix];
		double uy= vy[e*nxyz+iz*lxy+iy*lx1+ix];
		double uz= vz[e*nxyz+iz*lxy+iy*lx1+ix];
		double temp = t [ e*nxyz+iz*lxy+iy*lx1+ix];
		int ips;
		double ps[10]; // ps is size of ldimt which is 3. Not sure npscal is also 3. Need to check with Dr.Tania
		for (ips=0;ips<npscal;ips++){
			ps[ips]=t[(ips+1)*nlel+e*nxyz+iz*lxy+iy*lx1+ix]; // 5 th dimension of t is idlmt which is 3. Not sure how the  nekasgn access ips+1. Need to check with Dr.Tania
		}
		double pa = pr [e*nxyz+iz*lxy+iy*lx1+ix];
		double p0= p0th;
		double si2 =  sii[e*nxyz+iz*lxy+iy*lx1+ix];
		double si3 =  siii[e*nxyz+iz*lxy+iy*lx1+ix];
		double udiff =  vdiff[(ifield-1)*nlel+e*nxyz+iz*lxy+iy*lx1+ix];
		double utrans =  vtrans[(ifield-1)*nlel+e*nxyz+iz*lxy+iy*lx1+ix];
		char cbu1 = cb[0];
		char cbu2 = cb[1];
		char cbu3 = cb[2];
		//userf(i,j,k,eg)
		int egg=gllel[eg];// use glleg twice. check with Dr.Tania to find the real effect. If it reverses can use just id for the following arrays indices. adeesha.
		double ffx,ffy,ffz,qvol;  //actually these things should copy back to the cpu varibales. adeesha.
		if (two_way >=2) {
			if (istep > time_delay) {
				ffx =  ptw[eg*nxyz+iz*lxy+iy*lx1+ix]/vtrans[eg*nxyz+iz*lxy+iy*lx1+ix] /(1.0-ptw[3*lxyzlelt+eg*nxyz+iz*lx1*ly1+iy*lx1+ix]);
				ffy =  ptw[1*lxyzlelt+eg*lx1*nxyz+iz*lxy+iy*lx1+ix]/vtrans[eg*nxyz+iz*lxy+iy*lx1+ix] /(1.0-ptw[3*lxyzlelt+eg*nxyz+iz*lxy+iy*lx1+ix]);
				ffz =  ptw[2*lxyzlelt+eg*nxyz+iz*lx1*ly1+iy*lx1+ix]/vtrans[eg*nxyz+iz*lxy+iy*lx1+ix] /(1.0-ptw[3*lxyzlelt+eg*nxyz+iz*lxy+iy*lx1+ix]);
				if (icmtp == 1){
					qvol= ptw[4*lxyzlelt+eg*nxyz+iz*lx1*ly1+iy*lx1+ix] + rhs_fluidp[4*lxyzlelt+eg*nxyz+iz*lxy+iy*lx1+ix];
				}
				else{
					qvol=0.0;
				}
			}
			else{
				ffx = 0.0;
				ffy = 0.0;
				ffz = 0.0;
			}

		}
		else{
			ffx = 0.0;
			ffy = 0.0;
			ffz = 0.0;

		}			

		usrf[1*nxyz+iz*(lxy)+iy*lx1+ix] = ffx*u[e*toteqlxyz+iz*(lxy)+iy*lx1+ix]*phig[id];
		usrf[2*nxyz+iz*(lxy)+iy*lx1+ix] = ffy*u[e*toteqlxyz+iz*(lxy)+iy*lx1+ix]*phig[id];
		usrf[3*nxyz+iz*(lxy)+iy*lx1+ix] = ffz*u[e*toteqlxyz+iz*(lxy)+iy*lx1+ix]*phig[id];
		usrf[4*nxyz+iz*(lxy)+iy*lx1+ix] = qvol;




	}
}

extern "C" void cmtusrf_gpu_wrapper_(int *glbblockSize1,double *d_usrf,double *d_xm1,double *d_ym1,double *d_zm1,double *d_vx,double *d_vy,double *d_vz,double *d_t,double *d_pr,double *d_sii,double *d_siii,double *d_vdiff,double *d_vtrans,char *d_cb,double *d_ptw,int *d_lglel,int *d_gllel,double *d_rhs_fluidp,double *d_u,double *d_phig,int *lx1,int *ly1,int *lz1,int *toteq,int  *istep,int  *npscal,int *two_way,int *time_delay,int *icmtp,int *nelt, int *lelt,double *p0th,int *ifield){

	hipDeviceSynchronize();
	hipError_t code1 = hipPeekAtLastError();

	printf("CUDA: Start cmtusrf_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));

	printf("CUDA: Start cmtusrf_gpu_wrapper values lx1=%d,ly1=%d,lz1=%d,toteq=%d,istep=%d,npscal=%d,two_way=%d,time_delay=%d,icmtp=%d,nelt=%d,lelt=%d,p0th=%lf,ifield=%d\n",lx1[0],ly1[0],lz1[0],toteq[0],istep[0],npscal[0],two_way[0],time_delay[0],icmtp[0],nelt[0],lelt[0],p0th[0],ifield[0]);

	int lxy = lx1[0]*ly1[0];
	int nxyz = lxy*lz1[0];
	int lxyzlelt = nxyz*lelt[0];
	int nnel = nxyz*nelt[0];
	int nlel=nxyz*lelt[0];
	int toteqlxyz=toteq[0]*nxyz;

	int blockSize = glbblockSize1[0], gridSize;
	gridSize = (int)ceil((float)nnel/blockSize);
	cmtusrf_gpu_kernel<<<gridSize, blockSize>>>(d_usrf,d_xm1,d_ym1,d_zm1,d_vx,d_vy,d_vz,d_t,d_pr,d_sii,d_siii,d_vdiff,d_vtrans,d_cb,d_ptw,d_lglel,d_gllel,d_rhs_fluidp,d_u,d_phig,nnel,lx1[0],ly1[0],lz1[0],lxy,nxyz,lxyzlelt,toteqlxyz,istep[0],npscal[0],two_way[0],time_delay[0],icmtp[0],nlel,p0th[0],ifield[0]);

	hipError_t code2 = hipPeekAtLastError();

	printf("CUDA: End cmtusrf_gpu_wrapper cuda status: %s\n",hipGetErrorString(code2));
}

__global__ void compute_gradients_gpu_kernel1(double *ud, double *u, double *phig, int nnel,int lx1,int ly1,int lz1, int lxy,int nxyz,int toteqlxyz,int eq){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nnel){
		int ix = id % lx1;
		int iy = (id/lx1)%ly1;
		int iz = (id / (lxy))%lz1;
		int e = id/nxyz;

		ud[id] = u[e*toteqlxyz+eq*nxyz+iz*lxy+iy*lx1+ix]/phig[id];

	}
}

__global__ void compute_gradients_gpu_kernel2(double *ur,double *us,double *ut,double *gradu, double *jacmi,double *rxm1,double *rym1, double *rzm1, double *sxm1, double *sym1, double *szm1,double *txm1, double *tym1, double *tzm1, int lx1, int ly1, int lz1, int lxy, int nxyz, int toteqlxyz,int eq,int nnel){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nnel){
		int ix = id % lx1;
		int iy = (id/lx1)%ly1;
		int iz = (id / (lxy))%lz1;
		int e = id/nxyz;

		gradu[0*toteqlxyz+eq*nxyz+id] =  jacmi[id] *( rxm1[id]*ur[id]+ sxm1[id]*us[id]+txm1[id]*ut[id]);
		gradu[1*toteqlxyz+eq*nxyz+id] =  jacmi[id] *( rym1[id]*ur[id]+ sym1[id]*us[id]+tym1[id]*ut[id]);
		gradu[2*toteqlxyz+eq*nxyz+id] =  jacmi[id] *( rzm1[id]*ur[id]+ szm1[id]*us[id]+tzm1[id]*ut[id]);

	}
}

__global__ void compute_gradients_gpu_kernel3(double *ur,double *us,double *gradu, double *jacmi,double *rxm1, double *rym1, double *sxm1, double *sym1, int nnel,int lx1, int ly1, int lxy,int nxyz, int toteqlxyz, int eq,int lz1){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nnel){
		int ix = id % lx1;
		int iy = (id/lx1)%ly1;
		int iz = (id / (lxy))%lz1;
		int e = id/nxyz;

		gradu[0*toteqlxyz+eq*nxyz+id] =  jacmi[id] *( rxm1[id]*ur[id]+ sxm1[id]*us[id]);
		gradu[1*toteqlxyz+eq*nxyz+id] =  jacmi[id] *( rym1[id]*ur[id]+ sym1[id]*us[id]);

	}
}


// original  compute_gradients is in intpdiff.f file
extern "C" void compute_gradients_gpu_wrapper_(int *glbblockSize1,double *d_u,double *d_phig, double *d_dxm1,double *d_dxtm1, double *d_gradu,double *d_jacmi,double *d_rxm1,double *d_rym1,double *d_rzm1,double *d_sxm1,double *d_sym1,double *d_szm1,double *d_txm1,double *d_tym1,double *d_tzm1,int *lx1, int *ly1, int *lz1, int *nelt, int *lelt,int *toteq,int *lxd,int *lyd,int *lzd,int *if3d){

	hipDeviceSynchronize();
	hipError_t code1 = hipPeekAtLastError();

	printf("CUDA: Start compute_gradients_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));

	printf("CUDA: Start compute_gradients_gpu_wrapper values lx1,ly1,lz1,nelt,lelt,toteq,lxd,lyd,lzd,if3d\n",lx1[0],ly1[0],lz1[0],nelt[0],lelt[0],toteq[0],lxd[0],lyd[0],lzd[0],if3d[0]);

	int nnel = lx1[0]*ly1[0]*lz1[0]*nelt[0];

	int lxy  = lx1[0]*ly1[0];
	int  lyz  = ly1[0]*lz1[0];
	int nxyz = lxy*lz1[0];
	int m0 = lx1[0]-1;
	int ndlel=lxd[0]*lyd[0]*lzd[0]*lelt[0];
	int toteqlxyz= nxyz*toteq[0];

	double *d_ur;
	double *d_us;
	double *d_ut;
	double *d_ud;

	hipMalloc((void**)&d_us,ndlel*sizeof(double));
	hipMalloc((void**)&d_ut,ndlel*sizeof(double));
	hipMalloc((void**)&d_ur,ndlel*sizeof(double));
	hipMalloc((void**)&d_ud,ndlel*sizeof(double));
	hipMemset(d_ur, 0.0, ndlel*sizeof(double));
	hipMemset(d_us, 0.0, ndlel*sizeof(double));
	hipMemset(d_ut, 0.0, ndlel*sizeof(double));
	hipMemset(d_ud, 0.0, ndlel*sizeof(double));


	int blockSize = glbblockSize1[0], gridSize;
	gridSize = (int)ceil((float)nnel/blockSize);

	for(int eq=0; eq<toteq[0];eq++){


		compute_gradients_gpu_kernel1<<<gridSize, blockSize>>>(d_ud,d_u,d_phig,nnel,lx1[0],ly1[0],lz1[0],lxy,nxyz,toteqlxyz,eq);

		if(if3d[0]){
			gpu_local_grad3(d_ur,d_us,d_ut,d_ud,m0,1,d_dxm1,d_dxtm1,nelt[0]);// why define  d_ur .. to ldd if only using lx1. check with Dr.Tania. adeesha.
			compute_gradients_gpu_kernel2<<<gridSize, blockSize>>>(d_ur,d_us,d_ut,d_gradu,d_jacmi,d_rxm1,d_rym1,d_rzm1,d_sxm1,d_sym1,d_szm1,d_txm1,d_tym1,d_tzm1,lx1[0],ly1[0],lz1[0],lxy,nxyz,toteqlxyz,eq,nnel);

		}
		else{
			gpu_local_grad2(d_ur,d_us,d_ud,m0,1,d_dxm1,d_dxtm1,nelt[0]);
			compute_gradients_gpu_kernel3<<<gridSize, blockSize>>>(d_ur,d_us,d_gradu,d_jacmi,d_rxm1,d_rym1,d_sxm1,d_sym1,nnel,lx1[0],ly1[0],lxy,nxyz,toteqlxyz,eq,lz1[0]);


		}	
	}
	hipFree(d_ur);
	hipFree(d_us);
	hipFree(d_ut);
	hipFree(d_ud);

	hipError_t code2 = hipPeekAtLastError();

	printf("CUDA: End compute_gradients_wrapper cuda status: %s\n",hipGetErrorString(code2));

}

__global__ void convective_cmt_gpu_kernel1(double *convh,double *vxd,double *vyd,double *vzd,double *ju1,double *ju2,int eq, int ndlel,int if3d){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<ndlel){
		convh[ndlel+id] = convh[id];
		convh[2*ndlel+id] = convh[id];
		convh[id]=convh[id]*vxd[id];
		convh[ndlel+id]=convh[ndlel+id]*vyd[id];
		if(if3d){
			convh[2*ndlel+id]=convh[2*ndlel+id]*vzd[id];
		}
		convh[2*(eq-1)+id]=convh[2*(eq-1)+id]+ju1[id]*ju2[id]; // works only when toteq <=5. Otherwise eq-1 will be larger than ldim

	}
}

__global__ void convective_cmt_gpu_kernel2(double *convh,double *vxd,double *vyd,double *vzd,double *ju1,double *ju2,int eq, int ndlel){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<ndlel){

		convh[id]=convh[id]+ju1[id]*ju2[id]; // works only when toteq <=5. Otherwise eq-1 will be larger than ldim
		convh[ndlel+id] = convh[id];
		convh[2*ndlel+id] = convh[id];
		convh[id]=convh[id]*vxd[id];
		convh[ndlel+id]=convh[ndlel+id]*vyd[id];
		convh[2*ndlel+id]=convh[2*ndlel+id]*vzd[id];

	}
}

__global__ void convective_cmt_flux_div_integral_dealiased_gpu_kernel1(double *totalh,double *rx,double *ur,double *us,double *ut,int lxd, int lyd, int lzd, int lxyd,  int lxyzd, int lxyzdldimldim,int ndlel,int if3d){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<ndlel){
		int ix = id % lxd;
		int iy = (id/lxd)%lyd;
		int iz = (id / (lxyd))%lzd;
		int e = id/lxyzd;
		ur[id]=ur[id]+totalh[0*lxyzd+id]*rx[e*lxyzdldimldim+0*lxd*lyd*lzd+iz*lxd*lyd+iy*lxd+ix]; // this rx seems to be collection of rxm1 to tzm1. all 9 of them. so this should be  changed to seperate arrays. but original version is  implemented in this way.  need to check with Dr.Tania. adeesha.
		ur[id]=ur[id]+totalh[1*lxyzd+id]*rx[e*lxyzdldimldim+1*lxyzd+iz*lxyd+iy*lxd+ix];
		ur[id]=ur[id]+totalh[2*lxyzd+id]*rx[e*lxyzdldimldim+2*lxyzd+iz*lxyd+iy*lxd+ix];

		us[id]=us[id]+totalh[0*lxyzd+id]*rx[e*lxyzdldimldim+3*lxyzd+iz*lxyd+iy*lxd+ix];
		us[id]=us[id]+totalh[1*lxyzd+id]*rx[e*lxyzdldimldim+4*lxyzd+iz*lxyd+iy*lxd+ix];
		us[id]=us[id]+totalh[2*lxyzd+id]*rx[e*lxyzdldimldim+5*lxyzd+iz*lxyd+iy*lxd+ix];

		if(if3d){
			ut[id]=ut[id]+totalh[0*lxyzd+id]*rx[e*lxyzdldimldim+6*lxyzd+iz*lxyd+iy*lxd+ix];
			ut[id]=ut[id]+totalh[1*lxyzd+id]*rx[e*lxyzdldimldim+7*lxyzd+iz*lxyd+iy*lxd+ix];
			ut[id]=ut[id]+totalh[2*lxyzd+id]*rx[e*lxyzdldimldim+8*lxyzd+iz*lxyd+iy*lxd+ix];
		}
	}
}

__global__ void convective_cmt_evaluate_aliased_conv_h_gpu_kernel1(double *ju1,double *ju2,double *phig,double *pr, double *convh, double *u,double *totalh,int lxd,int lyd,int lzd,int lxyd,int lxyzd,int lxyzdlelt,int lx1,int ly1,int lz1,int lxy,int lxyz,int toteqlxyz,int ndlel,int eq,int ldim,double *vxd,  double *vyd,double *vzd,int if3d){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<ndlel){
		int ix = id % lxd;
		int iy = (id/lxd)%lyd;
		int iz = (id / (lxyd))%lzd;
		int e = id/lxyzd;
		ju1[id]=phig[id];
		ju2[id]=pr[id];
		if(eq<4){
			convh[id]=u[e*toteqlxyz+eq*lxyz+iz*lxy+iy*lx1+ix];				     for(int j=1;j<ldim;j++){
				convh[j*lxyzdlelt+id]=convh[id];
			}
			convh[id]=convh[id]*vxd[id];
			convh[lxyzdlelt+id]=convh[lxyzdlelt+id]*vyd[id];
			if(if3d){
				convh[2*lxyzdlelt+id]=convh[2*lxyzdlelt+id]*vzd[id];
			}
			if(eq>0){
				convh[(eq-1)*lxyzdlelt+id] = convh[(eq-1)*lxyzdlelt+id]+ju1[id]*ju2[id];	
			}
		}
		else if(eq==4){
			convh[id]=u[e*toteqlxyz+eq*lxyz+iz*lxy+iy*lx1+ix];
			convh[id] = convh[id]+ju1[id]*ju2[id];
			for(int j=1;j<ldim;j++){
				convh[j*lxyzdlelt+id]=convh[id];
			}
			convh[id]=convh[id]*vxd[id];
			convh[lxyzdlelt+id]=convh[lxyzdlelt+id]*vyd[id];
			convh[2*lxyzdlelt+id]=convh[2*lxyzdlelt+id]*vzd[id];

		}
		else{
			//send error message back to the fortran program.
		}

		totalh[id]=convh[id];
		totalh[lxyzdlelt+id]=convh[lxyzdlelt+id];
		totalh[2*lxyzdlelt+id]=convh[2*lxyzdlelt+id];



	}
}


extern "C" void convective_cmt_gpu_wrapper_(int *glbblockSize1,int *glbblockSize2,double *d_wkd,double *d_convh,double *d_vxd,double *d_vyd,double *d_vzd,double *d_totalh,double *d_rx,double *d_dg,double *d_dgt,double *d_res1,int *lx1, int *ly1, int *lz1, int *nelt, int *lelt,int *toteq, int *lxd, int *lyd, int *lzd, int *ldim,int *if3d,double *d_u,double *d_phig,double *d_pr ){

	hipDeviceSynchronize();
	hipError_t code1 = hipPeekAtLastError();

	printf("CUDA: Start convective_cmt_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));

	printf("CUDA: Start convective_cmt_gpu_wrapper values lx1=%d ,ly1=%d,lz1=%d,nelt=%d,lelt=%d,toteq=%d,lxd=%d,lyd=%d, lzd=%d,ldim=%d,if3d=%d \n", lx1[0],ly1[0],lz1[0],nelt[0],lelt[0],toteq[0],lxd[0],lyd[0], lzd[0],ldim[0],if3d[0]);

	int nnel = lx1[0]*ly1[0]*lz1[0]*nelt[0];

	int lxy  = lx1[0]*ly1[0];
	int  lxyz  = lxy*lz1[0];
	int toteqlxyz= lxyz*toteq[0];
	int m0 = lx1[0]-1;
	int ndlel=lxd[0]*lyd[0]*lzd[0]*lelt[0];
	int ld = 2*lxd[0];
	int ldd=lxd[0]*lyd[0]*lzd[0];
	int ldw=2*pow(ld,ldim[0]);

	int lxyd= lxd[0]*lyd[0];
	int lxyzd= lxyd*lzd[0];
	int lxyzdlelt=lxyzd*lelt[0];
	int lxyzdldimldim= lxyzd*ldim[0]*ldim[0];

	double *d_ju1;
	double *d_ju2;
	double *d_jgl;
	double *d_jgt;
	double *d_w;


	double *d_ur;
	double *d_us;
	double *d_ut;
	double *d_ud;
	double *d_tu;

	int *d_pjgl;
	int *d_pdg;

	hipMalloc((void**)&d_ju1,nelt[0]*ldd*sizeof(double));
	hipMalloc((void**)&d_ju2,nelt[0]*ldd*sizeof(double));
	hipMalloc((void**)&d_jgl,ldd*sizeof(double));// No nelt[0] here. Need to talk with Mohamed to find the reason or Talk with Dr.Tania. adeesha
	hipMalloc((void**)&d_jgt,ldd*sizeof(double)); // same as above
	hipMalloc(&d_w, nelt[0]*ldw*sizeof(double));


	hipMalloc((void**)&d_ur,nelt[0]*ldd*sizeof(double));
	hipMalloc((void**)&d_us,nelt[0]*ldd*sizeof(double));
	hipMalloc((void**)&d_ut,nelt[0]*ldd*sizeof(double));
	hipMalloc((void**)&d_ud,nelt[0]*ldd*sizeof(double));
	hipMalloc((void**)&d_tu,nelt[0]*ldd*sizeof(double));

	hipMalloc((void**)&d_pjgl,nelt[0]*2*lxd[0]*sizeof(int));
	hipMalloc((void**)&d_pdg,nelt[0]*2*lxd[0]*sizeof(int));

	hipMemset(d_ju1, 0.0, nelt[0]*ldd*sizeof(double));
	hipMemset(d_ju2, 0.0, nelt[0]*ldd*sizeof(double));
	hipMemset(d_jgl, 0.0, ldd*sizeof(double));// May need to copy from cpu to gpu. check with Dr.Tania. adeesha. 
	hipMemset(d_jgt, 0.0, ldd*sizeof(double));


	int blockSize = glbblockSize1[0], gridSize;
	gridSize = (int)ceil((float)nnel/blockSize);

	int i=0;

	//printf("GPU :eqnsolver.cu : convective_cmt_gpu_wrapper after malloc\n");

	for(int eq=0; eq<toteq[0];eq++){

		if(lxd[0]>lx1[0]){
			//evaluate_dealiased_conv_h(e,eq)
			//printf("GPU :eqnsolver.cu : convective_cmt_gpu_wrapper start if lxd?lx1\n");
			if(eq==0){
				for(int j=0;j<ldim[0];j++){
					//intp_rstd(convh(1,j),u(1,1,1,eq+j,e),lx1,lxd,if3d,0)
					// call get_int_ptr (i,mx,md) // a function that goes very deep. Try to  do something about this. Talk with Dr.Tania. adeesha. this function is important because it fills jgt and jgl.
					//printf("GPU :eqnsolver.cu : convective_cmt_gpu_wrapper start if lxd>lx1 before gpu_gett_int_ptr\n");
					//gpu_get_int_ptr(&i,if3d[0], lx1[0], lxd[0], nelt[0],d_jgl, d_jgt,d_wkd,lxd[0],d_pjgl);
					//printf("GPU :eqnsolver.cu : convective_cmt_gpu_wrapper start if lxd>lx1 after gpu_gett_int_ptr\n");
					gpu_specmpn(d_convh+j*ldd, lxd[0], d_u+(j+1)*lxyz ,lx1[0], d_jgl, d_jgt, if3d[0], d_w, ldw, nelt[0], toteq[0], j+1, true ); // this may have toteq in aSize. which is confusing. Talk with Mohamed and Dr.Tania about this. adeesha.
					//also check d_u(j+1) or d_u(j+1+eq).  adeesha.
					hipDeviceSynchronize();
					code1 = hipPeekAtLastError();
					printf("CUDA: eq=0 and j=%d convective_cmt_gpu_wrapper after gpu_specmpn cuda status: %s\n",j,hipGetErrorString(code1));
				}
				//printf("GPU :eqnsolver.cu : convective_cmt_gpu_wrapper end if  eq==0 toteq= %d\n",eq);
			}
			else{
				//printf("GPU :eqnsolver.cu : convective_cmt_gpu_wrapper start of else  eq==0 toteq= %d\n",eq);
				//gpu_get_int_ptr(&i,if3d[0], lx1[0], lxd[0], nelt[0],d_jgl, d_jgt,d_wkd,lxd[0],d_pjgl);

				gpu_specmpn(d_ju1, lxd[0], d_phig, lx1[0], d_jgl, d_jgt, if3d, d_w, ldw, nelt[0], 1, 0,true);
				//gpu_get_int_ptr(&i,if3d[0], lx1[0], lxd[0], nelt[0],d_jgl, d_jgt,d_wkd,lxd[0],d_pjgl);
				hipDeviceSynchronize();
				code1 = hipPeekAtLastError();
				printf("CUDA: eq!=0 convective_cmt_gpu_wrapper after gpu_specmpn with d_ju1 cuda status: %s\n",hipGetErrorString(code1));


				gpu_specmpn(d_ju2, lxd[0], d_pr, lx1[0], d_jgl, d_jgt, if3d, d_w, ldw, nelt[0], 1, 0,true);
				hipDeviceSynchronize();
				code1 = hipPeekAtLastError();
				printf("CUDA: eq!=0 convective_cmt_gpu_wrapper after gpu_specmpn with d_ju2 cuda status: %s\n",hipGetErrorString(code1));


				if(eq<4){
					//gpu_get_int_ptr(&i,if3d[0], lx1[0], lxd[0], nelt[0],d_jgl, d_jgt,d_wkd,lxd[0],d_pjgl);

					gpu_specmpn(d_convh, lxd[0], d_u+eq*lxyz,lx1[0], d_jgl, d_jgt, if3d[0], d_w, ldw, nelt[0], toteq[0], eq,true);
					int blockSize, gridSize;

					// Number of threads in each thread block
					blockSize = glbblockSize1[0];
					gridSize = (int)ceil((float)ndlel/blockSize);
					convective_cmt_gpu_kernel1<<<gridSize, blockSize>>>(d_convh,d_vxd,d_vyd,d_vzd,d_ju1,d_ju2,eq,ndlel,if3d[0]);
				}	
				else if(eq==4){
					//gpu_get_int_ptr(&i,if3d[0], lx1[0], lxd[0], nelt[0],d_jgl, d_jgt,d_wkd,lxd[0],d_pjgl);

					gpu_specmpn(d_convh, lxd[0], d_u+eq*lxyz, lx1[0], d_jgl, d_jgt, if3d[0], d_w, ldw, nelt[0],toteq[0],eq,true);
					hipDeviceSynchronize();
					code1 = hipPeekAtLastError();
					printf("CUDA: convective_cmt_gpu_wrapper eq==4 else after gpu_specmpn cuda status: %s\n",hipGetErrorString(code1));

					convective_cmt_gpu_kernel2<<<gridSize, blockSize>>>(d_convh,d_vxd,d_vyd,d_vzd,d_ju1,d_ju2,eq,ndlel);
					hipDeviceSynchronize();
					code1 = hipPeekAtLastError();
					printf("CUDA: convective_cmt_gpu_wrapper eq==4 else after convective_cmt_gpu_kernel2 cuda status: %s\n",hipGetErrorString(code1));



				}


			}

			hipDeviceSynchronize();
			code1 = hipPeekAtLastError();
			printf("CUDA: convective_cmt_gpu_wrapper after first if cuda status: %s\n",hipGetErrorString(code1));

			//			printf("GPU :eqnsolver.cu : convective_cmt_gpu_wrapper after if eq before else toteq= %d\n",eq);
			gpu_double_copy_gpu_wrapper(glbblockSize2[0],d_totalh,0,d_convh,0,3*ldd);//this is calling an extern functions. Check how this works. adeesha
			//flux_div_integral_dealiased(e,eq)

			hipMemset(d_ur, 0.0, nelt[0]*ldd*sizeof(double));
			hipMemset(d_us, 0.0, nelt[0]*ldd*sizeof(double));
			hipMemset(d_ut, 0.0, nelt[0]*ldd*sizeof(double));
			hipMemset(d_ud, 0.0, nelt[0]*ldd*sizeof(double));
			hipMemset(d_tu, 0.0, nelt[0]*ldd*sizeof(double));
			//call get_dgl_ptr(ip,lxd,lxd) ! fills dg, dgt  need to implement this function somehow. adeesha.
			int ip=0;
			//gpu_get_dgl_ptr ( &ip,if3d[0], lx1[0],lxd[0], nelt[0],d_jgl, d_jgt,d_wkd,lxd[0],d_pdg);
			blockSize = glbblockSize1[0];
			gridSize = (int)ceil((float)ndlel/blockSize);
			convective_cmt_flux_div_integral_dealiased_gpu_kernel1<<<gridSize, blockSize>>>(d_totalh,d_rx,d_ur,d_us,d_ut,lxd[0],lyd[0],lzd[0],lxyd,lxyzd,lxyzdldimldim,ndlel,if3d[0]);

			hipDeviceSynchronize();
			code1 = hipPeekAtLastError();
			printf("CUDA: convective_cmt_gpu_wrapper after convective_cmt_flux_div_integral_dealiased_gpu_kernel1 cuda status: %s\n",hipGetErrorString(code1));

			if(if3d){
				//uncooment after fix gpu_local_grad3_t
				gpu_local_grad3_t(d_ud, d_ur, d_us, d_ut, lxd[0], d_dg, d_dgt, d_w, nelt[0]);
			}
			else{
				gpu_local_grad2_t(d_ud, d_ur, d_us, lxd[0], d_dg, d_dgt, d_w, nelt[0]);

			}
			hipDeviceSynchronize();
			code1 = hipPeekAtLastError();
			printf("CUDA: convective_cmt_gpu_wrapper after gpu_local_grad_kernel1 cuda status: %s\n",hipGetErrorString(code1));

			//gpu_get_int_ptr(&i,if3d[0], lx1[0], lxd[0], nelt[0],d_jgl, d_jgt,d_wkd,lxd[0],d_pjgl);
			gpu_specmpn(d_tu,lx1[0],d_ud,lxd[0],d_jgt,d_jgl,if3d[0],d_w,ldw,nelt[0],1,0,false);
			gpu_neksub2(glbblockSize2[0],d_res1+eq*nnel,d_tu,nnel);
			hipDeviceSynchronize();
			code1 = hipPeekAtLastError();
			printf("CUDA: convective_cmt_gpu_wrapper end of lxd>lx1 if_gpu_kernel1 cuda status: %s\n",hipGetErrorString(code1));

		}


		else{
			printf("GPU :eqnsolver.cu : convective_cmt_gpu_wrapper after if lxd >lx1 before else toteq= %d\n",eq);
			//evaluate_aliased_conv_h(e,eq)
			blockSize = glbblockSize1[0];
			gridSize = (int)ceil((float)ndlel/blockSize);
			convective_cmt_evaluate_aliased_conv_h_gpu_kernel1<<<gridSize, blockSize>>>(d_ju1,d_ju2,d_phig,d_pr,d_convh,d_u,d_totalh,lxd[0],lyd[0],lzd[0],lxyd,lxyzd,lxyzdlelt,lx1[0],ly1[0], lz1[0],lxy,lxyz,toteqlxyz,ndlel,eq,ldim[0],d_vxd,d_vyd,d_vzd,if3d[0]);

			hipMemset(d_ur, 0.0, nelt[0]*ldd*sizeof(double));
			hipMemset(d_us, 0.0, nelt[0]*ldd*sizeof(double));
			hipMemset(d_ut, 0.0, nelt[0]*ldd*sizeof(double));
			hipMemset(d_ud, 0.0, nelt[0]*ldd*sizeof(double));
			hipMemset(d_tu, 0.0, nelt[0]*ldd*sizeof(double));
			// implement the missing part.  adeesha.
			if(if3d){
				//uncomment after fix gpu_local_grad3_t
				gpu_local_grad3_t(d_ud, d_ur, d_us, d_ut, lxd[0], d_dg, d_dgt, d_w, nelt[0]);
			}
			else{
				gpu_local_grad2_t(d_ud, d_ur, d_us, lxd[0], d_dg, d_dgt, d_w, nelt[0]);

			}
			gpu_neksub2(glbblockSize2[0],d_res1+eq*nnel,d_tu,nnel);

		}
		hipDeviceSynchronize();
		code1 = hipPeekAtLastError();
		printf("CUDA: End eq for loop eq = %d convective_cmt_wrapper cuda status: %s\n",eq,hipGetErrorString(code1));
	}
	hipFree(d_ur);
	hipFree(d_us);
	hipFree(d_ut);
	hipFree(d_ud);
	hipFree(d_tu);

	hipFree(d_ju1);
	hipFree(d_ju2);
	hipFree(d_jgl);
	hipFree(d_jgt);
	hipFree(d_w);

	hipFree(d_pjgl);
	hipFree(d_pdg);
	hipDeviceSynchronize();
	hipError_t code2 = hipPeekAtLastError();
	printf("CUDA: End convective_cmt_wrapper cuda status: %s\n",hipGetErrorString(code2));
}

__global__ void viscous_cmt_gpu_kernel1(double *diffh,double *gradu, double *vtrans, double *vdiff, double * vx, double *vy, double *vz, double *u, double *viscscr, int lx1,int ly1,int lz1, int lxy, int nxyz,  int toteq, int nnel, int lxyzlelt, int toteqlxyz, int toteqlxyzlelt, int irho, int ilam, int imu, int icv, int iknd, int inus,int eq,int  if3d,int ldim){

	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nnel){

		int ix = id % lx1;
		int iy = (id/lx1)%ly1;
		int iz = (id / (lxy))%lz1;
		int e = id/nxyz;

		//agradu(diffh,gradu,e,eq) // flux =diffh change later. adeesha
		// call fluxj_ns (flux,du,e,eq)
		if(eq < toteq-1){
			if(eq==1){
				//A21kldUldxk(flux(1,1),gradu,e)
				double dU1x = gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU2x=gradu[0*toteqlxyzlelt+1*lxyzlelt+id ];
				double  dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU3y=gradu[1*toteqlxyzlelt+2*lxyzlelt+id ];
				double  dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU4z=gradu[2*toteqlxyzlelt+3*lxyzlelt+id ];
				double  rho   =vtrans[(irho-1)*lxyzlelt+id];
				double  lambda=vdiff[(ilam-1)*lxyzlelt+id];
				double  mu    =vdiff[(imu-1)*lxyzlelt+id];
				double  u1    =vx[id];
				double  u2    =vy[id];
				double  u3    =vz[id];
				double  lambdamu=lambda+2.0*mu;
				diffh[0*lxyzlelt+id]=(lambda*(dU4z+dU3y-u3*dU1z-u2*dU1y)+lambdamu*(dU2x-u1*dU1x))/rho;

				//A22kldUldxk(flux(1,2),gradu,e)
				dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double   dU3x=gradu[0*toteqlxyzlelt+2*lxyzlelt+id ];
				dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double   dU2y=gradu[1*toteqlxyzlelt+1*lxyzlelt+id ];
				rho   =vtrans[(irho-1)*lxyzlelt+id];
				mu    =vdiff[(imu-1)*lxyzlelt+id];
				u1    =vx[id];
				u2    =vy[id];
				diffh[1*lxyzlelt+id]=mu*(dU3x+dU2y-u1*dU1y-u2*dU1x)/rho;

				//A23kldUldxk(flux(1,3),gradu,e)

				dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU4x=gradu[0*toteqlxyzlelt+3*lxyzlelt+id ];
				dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU2z=gradu[2*toteqlxyzlelt+1*lxyzlelt+id ];
				rho   =vtrans[(irho-1)*lxyzlelt+id];
				mu    =vdiff[(imu-1)*lxyzlelt+id];
				u1    =vx[id];
				u3    =vz[id];
				diffh[2*lxyzlelt+id]=mu*(dU4x+dU2z-u1*dU1z-u3*dU1x)/rho;

			}
			else if(eq==2){
				//A31kldUldxk(flux(1,1),gradu,e)
				double dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU3x=gradu[0*toteqlxyzlelt+2*lxyzlelt+id ];
				double dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU2y=gradu[1*toteqlxyzlelt+1*lxyzlelt+id ];
				double rho   =vtrans[(irho-1)*lxyzlelt+id];
				double mu    =vdiff[(imu-1)*lxyzlelt+id];
				double u1    =vx[id];
				double u2    =vy[id];
				diffh[0*lxyzlelt+id] = mu*(dU3x+dU2y-u1*dU1y-u2*dU1x)/rho;

				//A32kldUldxk(flux(1,2),gradu,e)
				dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU2x=gradu[0*toteqlxyzlelt+1*lxyzlelt+id ];
				dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU3y=gradu[1*toteqlxyzlelt+2*lxyzlelt+id ];
				double dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double   dU4z=gradu[2*toteqlxyzlelt+3*lxyzlelt+id ];
				rho   =vtrans[(irho-1)*lxyzlelt+id];;
				double lambda=vdiff[(ilam-1)*lxyzlelt+id];
				mu    =vdiff[(imu-1)*lxyzlelt+id];
				u1    =vx[id];
				u2    =vy[id];
				double u3    =vz[id];
				double lambdamu=lambda+2.0*mu;
				diffh[1*lxyzlelt+id]=(lambda*(dU4z+dU2x-u3*dU1z-u1*dU1x)+lambdamu*(dU3y-u2*dU1y))/rho;

				//A33kldUldxk(flux(1,3),gradu,e)
				dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU4y=gradu[1*toteqlxyzlelt+3*lxyzlelt+id ];
				dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double   dU3z=gradu[2*toteqlxyzlelt+2*lxyzlelt+id ];
				rho   =vtrans[(irho-1)*lxyzlelt+id];
				mu    =vdiff[(imu-1)*lxyzlelt+id];
				u2    =vy[id];
				u3    =vz[id];
				diffh[2*lxyzlelt+id]=mu*(dU4y+dU3z-u2*dU1z-u3*dU1y)/rho;	



			}
			else if(eq==3){
				//A41kldUldxk(flux(1,1),gradu,e)
				double dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU4x=gradu[0*toteqlxyzlelt+3*lxyzlelt+id ];
				double dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU2z=gradu[2*toteqlxyzlelt+1*lxyzlelt+id ];
				double rho   =vtrans[(irho-1)*lxyzlelt+id];
				double mu    =vdiff[(imu-1)*lxyzlelt+id];

				double u1    =vx[id];
				double u3    =vz[id];
				diffh[0*lxyzlelt+id]=mu*(dU4x+dU2z-u1*dU1z-u3*dU1x)/rho;

				//A42kldUldxk(flux(1,2),gradu,e)
				double dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU4y=gradu[1*toteqlxyzlelt+3*lxyzlelt+id ];
				dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU3z=gradu[2*toteqlxyzlelt+2*lxyzlelt+id ];
				rho   =vtrans[(irho-1)*lxyzlelt+id];
				mu    =vdiff[(imu-1)*lxyzlelt+id];

				double u2    =vy[id];
				u3    =vz[id];
				diffh[1*lxyzlelt+id]=mu*(dU4y+dU3z-u2*dU1z-u3*dU1y)/rho;

				//A43kldUldxk(flux(1,3),gradu,e)
				dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU2x=gradu[0*toteqlxyzlelt+1*lxyzlelt+id ];
				dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU3y=gradu[1*toteqlxyzlelt+2*lxyzlelt+id ];
				dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU4z=gradu[2*toteqlxyzlelt+3*lxyzlelt+id ];
				rho   =vtrans[(irho-1)*lxyzlelt+id];
				double lambda=vdiff[(ilam-1)*lxyzlelt+id];
				mu    =vdiff[(imu-1)*lxyzlelt+id];
				u1    =vx[id];
				u2    =vy[id];
				u3    =vz[id];
				double lambdamu=lambda+2.0*mu;
				diffh[2*lxyzlelt+id]=(lambda*(dU3y+dU2x-u2*dU1y-u1*dU1x)+lambdamu*(dU4z-u3*dU1z))/rho;	
			}

		}
		else{
			if(if3d){
				//a53kldUldxk(flux(1,3),gradu,e)
				double dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
				double dU2x=gradu[0*toteqlxyzlelt+1*lxyzlelt+id ];
				double  dU3x=gradu[0*toteqlxyzlelt+2*lxyzlelt+id ];
				double  dU4x=gradu[0*toteqlxyzlelt+3*lxyzlelt+id ];
				double  dU5x=gradu[0*toteqlxyzlelt+4*lxyzlelt+id ];
				double  dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU2y=gradu[1*toteqlxyzlelt+1*lxyzlelt+id ];
				double  dU3y=gradu[1*toteqlxyzlelt+2*lxyzlelt+id ];
				double  dU4y=gradu[1*toteqlxyzlelt+3*lxyzlelt+id ];
				double  dU5y=gradu[1*toteqlxyzlelt+4*lxyzlelt+id ];
				double dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
				double  dU2z=gradu[2*toteqlxyzlelt+1*lxyzlelt+id ];
				double  dU3z=gradu[2*toteqlxyzlelt+2*lxyzlelt+id ];
				double  dU4z=gradu[2*toteqlxyzlelt+3*lxyzlelt+id ];
				double  dU5z=gradu[2*toteqlxyzlelt+4*lxyzlelt+id ];
				double  rho   =vtrans[(irho-1)*lxyzlelt+id];
				double  cv    =vtrans[(icv-1)*lxyzlelt+id]/rho;
				double  lambda=vdiff[(ilam-1)*lxyzlelt+id];
				double  mu    =vdiff[(imu-1)*lxyzlelt+id];
				double K     =vdiff[(iknd-1)*lxyzlelt+id];;
				double  u1    =vx[id];
				double  u2    =vy[id];
				double  u3    =vz[id];
				double  E     =u[e*toteqlxyz+(toteq-1)*nxyz+iz*lxy+iy*lx1+ix]/rho;
				double lambdamu=lambda+mu;
				double kmcvmu=K-cv*mu;
				diffh[2*lxyzlelt+id]=(K*(dU5z-E*dU1z)+cv*u3*(lambda*dU4z+2*mu*dU4z+lambda*dU3y+lambda*dU2x)-K*u3*dU4z+cv*mu*u2*(dU4y+dU3z)+cv*mu*u1*(dU4x+dU2z)-K*u2*dU3z-K*u1*dU2z-cv*(lambda+2*mu)*u3*u3*dU1z+K*u3*u3*dU1z+ K*u2*u2*dU1z-cv*mu*u2*u2*dU1z+K*u1*u1*dU1z-cv*mu*u1*u1*dU1z-cv*(lambda+mu)*u2*u3*dU1y-cv*(lambda+mu)*u1*u3*dU1x)/(cv*rho);

			}
			else{
				for(int kfortoteq=0;kfortoteq<toteq;kfortoteq++){
					gradu[2*toteqlxyzlelt+kfortoteq*lxyzlelt+id]=0;
				}
				vz[id]=0;

			}


		}
		//a51kldUldxk(flux(1,1),gradu,e)
		double dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
		double dU2x=gradu[0*toteqlxyzlelt+1*lxyzlelt+id ];
		double  dU3x=gradu[0*toteqlxyzlelt+2*lxyzlelt+id ];
		double  dU4x=gradu[0*toteqlxyzlelt+3*lxyzlelt+id ];
		double  dU5x=gradu[0*toteqlxyzlelt+4*lxyzlelt+id ];
		double  dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
		double  dU2y=gradu[1*toteqlxyzlelt+1*lxyzlelt+id ];
		double  dU3y=gradu[1*toteqlxyzlelt+2*lxyzlelt+id ];
		double  dU4y=gradu[1*toteqlxyzlelt+3*lxyzlelt+id ];
		double  dU5y=gradu[1*toteq*toteqlxyzlelt+4*lxyzlelt+id ];
		double dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
		double  dU2z=gradu[2*toteqlxyzlelt+1*lxyzlelt+id ];
		double  dU3z=gradu[2*toteqlxyzlelt+2*lxyzlelt+id ];
		double  dU4z=gradu[2*toteqlxyzlelt+3*lxyzlelt+id ];
		double  dU5z=gradu[2*toteqlxyzlelt+4*lxyzlelt+id ];
		double  rho   =vtrans[(irho-1)*lxyzlelt+id];
		double  cv    =vtrans[(icv-1)*lxyzlelt+id]/rho;
		double  lambda=vdiff[(ilam-1)*lxyzlelt+id];
		double  mu    =vdiff[(imu-1)*lxyzlelt+id];
		double K     =vdiff[(iknd-1)*lxyzlelt+id];;
		double  u1    =vx[id];
		double  u2    =vy[id];
		double  u3    =vz[id];
		double  E     =u[e*toteqlxyz+(toteq-1)*nxyz+iz*lxy+iy*lx1+ix]/rho;
		double lambdamu=lambda+mu;
		double kmcvmu=K-cv*mu;
		diffh[0*lxyzlelt+id]=(K*dU5x+cv*lambda*u1*dU4z-kmcvmu*u3*dU4x+cv*lambda*u1*dU3y-kmcvmu*u2*dU3x+cv*mu*u3*dU2z+cv*mu*u2*dU2y+(cv*lambda-K+2*cv*mu)*u1*dU2x-cv*lambdamu*u1*u3*dU1z-cv*lambdamu*u1*u2*dU1y+(K*u3*u3-cv*mu*u3*u3+K*u2*u2-cv*mu*u2*u2-cv*lambda*u1*u1+K*u1*u1-2*cv*mu*u1*u1-E*K)*dU1x)/(cv*rho);

		//a52kldUldxk(flux(1,2),gradu,e)
		dU1x=gradu[0*toteqlxyzlelt+0*lxyzlelt+id ];
		dU2x=gradu[0*toteqlxyzlelt+1*lxyzlelt+id ];
		dU3x=gradu[0*toteqlxyzlelt+2*lxyzlelt+id ];
		dU4x=gradu[0*toteqlxyzlelt+3*lxyzlelt+id ];
		dU5x=gradu[0*toteqlxyzlelt+4*lxyzlelt+id ];
		dU1y=gradu[1*toteqlxyzlelt+0*lxyzlelt+id ];
		dU2y=gradu[1*toteqlxyzlelt+1*lxyzlelt+id ];
		dU3y=gradu[1*toteqlxyzlelt+2*lxyzlelt+id ];
		dU4y=gradu[1*toteqlxyzlelt+3*lxyzlelt+id ];
		dU5y=gradu[1*toteqlxyzlelt+4*lxyzlelt+id ];
		dU1z=gradu[2*toteqlxyzlelt+0*lxyzlelt+id ];
		dU2z=gradu[2*toteqlxyzlelt+1*lxyzlelt+id ];
		dU3z=gradu[2*toteqlxyzlelt+2*lxyzlelt+id ];
		dU4z=gradu[2*toteqlxyzlelt+3*lxyzlelt+id ];
		dU5z=gradu[2*toteqlxyzlelt+4*lxyzlelt+id ];
		rho   =vtrans[(irho-1)*lxyzlelt+id];
		cv    =vtrans[(icv-1)*lxyzlelt+id]/rho;
		lambda=vdiff[(ilam-1)*lxyzlelt+id];
		mu    =vdiff[(imu-1)*lxyzlelt+id];
		K     =vdiff[(iknd-1)*lxyzlelt+id];;
		u1    =vx[id];
		u2    =vy[id];
		u3    =vz[id];
		E     =u[e*toteqlxyz+(toteq-1)*nxyz+iz*lxy+iy*lx1+ix]/rho;
		lambdamu=lambda+mu;
		kmcvmu=K-cv*mu;
		diffh[1*lxyzlelt+id]=(K*dU5y+cv*lambda*u2*dU4z-kmcvmu*u3*dU4y+cv*mu*u3*dU3z+(cv*lambda-K+2*cv*mu)*u2*dU3y+cv*mu*u1*dU3x-kmcvmu*u1*dU2y+cv*lambda*u2*dU2x-cv*lambdamu*u2*u3*dU1z+(K*u3*u3-cv*mu*u3*u3-cv*lambda*u2*u2+K*u2*u2-2*cv*mu*u2*u2+K*u1*u1-cv*mu*u1*u1-E*K)*dU1y-cv*lambdamu*u1*u2*dU1x)/(cv*rho);


		//call fluxj_evm(flux,du,e,eq)

		if(eq==0){
			for(int jj=0;jj<ldim;jj++){
				diffh[jj*lxyzlelt+id]=  diffh[jj*lxyzlelt+id]+vdiff[(inus-1)*lxyzlelt+id]*gradu[jj*toteqlxyzlelt+id];
			}

		}
		else{
			if(eq<toteq-1){
				viscscr[id]=gradu[0*toteqlxyzlelt+(eq-1)*lxyzlelt+id ]; // problem with du indices. du(1,1,eq-1) third is for ldim check wih Dr.Tania adeesha.
				viscscr[id]=viscscr[id]*vdiff[(inus-1)*lxyzlelt+id];
				diffh[0*lxyzlelt+id]=  diffh[0*lxyzlelt+id]+viscscr[id]*vx[id]; 
				diffh[1*lxyzlelt+id]=  diffh[1*lxyzlelt+id]+viscscr[id]*vy[id];
				if(if3d){
					diffh[2*lxyzlelt+id]=  diffh[2*lxyzlelt+id]+viscscr[id]*vz[id];
				} 

			}
			else{
				if(if3d){
					viscscr[id] = vx[id]*vx[id]+vy[id]*vy[id]+vz[id]*vz[id];
				}
				else{
					viscscr[id] = vx[id]*vx[id]+vy[id]*vy[id];
				}
				viscscr[id] =  viscscr[id] *vdiff[(inus-1)*lxyzlelt+id];
				for(int jj=0;jj<ldim;jj++){
					diffh[jj*lxyzlelt+id]=  diffh[jj*lxyzlelt+id]+viscscr[id] *gradu[jj*toteqlxyzlelt+id];
				}
				for(int jj=0;jj<ldim;jj++){
					for(int eq2=1;eq2<ldim+1;eq2++){
						viscscr[id]=gradu[jj*toteqlxyzlelt+eq2*lxyzlelt+id ]* u[e*toteqlxyz+eq2*nxyz+iz*lxy+iy*lx1+ix]+vdiff[(inus-1)*lxyzlelt+id] ;
						viscscr[id]=viscscr[id]/vtrans[(irho-1)*lxyzlelt+id];	
						diffh[jj*lxyzlelt+id]=diffh[jj*lxyzlelt+id]-viscscr[id];
						diffh[jj*lxyzlelt+id]=  diffh[jj*lxyzlelt+id]+vdiff[(inus-1)*lxyzlelt+id] *gradu[jj*toteqlxyzlelt+(toteq-1)*lxyzlelt+id ];
					}

				} 



			}
		}
	}

}

__global__ void viscous_cmt_gpu_kernel2(double *graduf,double *normal,double *unx,double *uny, double *unz,double *iface_flux,double *hface,double *diffh,int ntot,int lxz2ldim,int lxz2ldimlelt,int lxyz,int eq,int lxyzlelt,double *area){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<ntot){
		int e = id/lxz2ldim;
		//diffh2graduf(e,eq,graduf)
		graduf[eq*lxz2ldimlelt+id]=0;
		for(int j=0;j<3;j++){
			if(j==1){normal[id]=unx[id];}
			if(j==2){normal[id]=uny[id];}
			if(j==3){normal[id]=unz[id];}
			int newi = iface_flux[id];
			hface[id] =diffh[j*lxyzlelt+e*lxyz+newi];
			graduf[eq*lxz2ldimlelt+id]=graduf[eq*lxz2ldimlelt+id]+hface[id]*normal[id];
		}
		graduf[eq*lxz2ldimlelt+id]= graduf[eq*lxz2ldimlelt+id]*area[id];
	}
}

__global__ void viscous_cmt_gpu_kernel3( double *ur, double *us, double *ut, double *jacmi, double *rxm1,double *rym1, double *rzm1, double *sxm1, double *sym1, double *szm1,double *txm1, double *tym1, double *tzm1,double *diffh,int ldim, int nnel,int lxyzlelt,double *bm1,int if3d){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nnel){
		//half_iku_cmt(res1(1,1,1,e,eq),diffh,e)
		//half_iku_cmt(res,diffh,e) 
		for(int j=0;j<ldim;j++){
			diffh[j*lxyzlelt+id]= diffh[j*lxyzlelt+id]*bm1[id];
		}
		//gradm11_t(res,diffh,const,e)	
		if(if3d){
			ur[id] =  jacmi[id] *( rxm1[id]*diffh[0*lxyzlelt+id]+ rym1[id]*diffh[1*lxyzlelt+id]+rzm1[id]*diffh[2*lxyzlelt+id]);
			us[id] =  jacmi[id] *( sxm1[id]*diffh[0*lxyzlelt+id]+ sym1[id]*diffh[1*lxyzlelt+id]+szm1[id]*diffh[2*lxyzlelt+id]);
			ut[id] =  jacmi[id] *( txm1[id]*diffh[0*lxyzlelt+id]+ tym1[id]*diffh[1*lxyzlelt+id]+tzm1[id]*diffh[2*lxyzlelt+id]);

		}
		else{
			ur[id] =  jacmi[id] *( rxm1[id]*diffh[0*lxyzlelt+id]+ rym1[id]*diffh[1*lxyzlelt+id]);
			us[id] =  jacmi[id] *( sxm1[id]*diffh[0*lxyzlelt+id]+sym1[id]*diffh[1*lxyzlelt+id]);
		}			
	}
}

__global__ void viscous_cmt_gpu_kernel4(double *res1,double *ud,int nnel, int  eq,int lxyzlelt){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nnel){
		ud[id]=ud[id]*1.0;
		res1[eq*lxyzlelt+id]=res1[eq*lxyzlelt+id]+ud[id];
	}

}


extern "C" void viscous_cmt_gpu_wrapper_(int *glbblockSize1,int *glbblockSize2,double *d_diffh,double *d_gradu,double *d_vtrans,double *d_vdiff,double *d_vx,double *d_vy,double *d_vz,double *d_u,double *d_viscscr,double *d_jacmi,double *d_rxm1,double *d_rym1,double *d_rzm1,double *d_sxm1,double *d_sym1,double *d_szm1,double *d_txm1,double *d_tym1,double *d_tzm1,double *d_graduf,double *d_unx,double *d_uny,double *d_unz,double *d_iface_flux,double *d_dxm1,double *d_dxtm1,double *d_res1,double *d_area,double *d_bm1,int *lx1,int *ly1,int *lz1,int *toteq,int *irho,int *ilam,int *imu,int *icv,int *iknd,int *inus,int *if3d,int *ldim,int *nelt,int *lelt){

	hipDeviceSynchronize();
	hipError_t code1 = hipPeekAtLastError();

	printf("CUDA: Start viscous_cmt_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));

	printf("CUDA: Start viscous_cmt_gpu_wrapper values lx1=%d,ly1=%d,lz1=%d,toteq=%d,irho=%d,ilam=%d,imu=%d,icv=%d,iknd=%d,inus=%d,if3d=%d,ldim=%d,nelt=%d,lelt=%d \n", lx1[0],ly1[0],lz1[0],toteq[0],irho[0],ilam[0],imu[0],icv[0],iknd[0],inus[0],if3d[0],ldim[0],nelt[0],lelt[0]);

	int lxy  = lx1[0]*ly1[0];
	int  lxz  = ly1[0]*lz1[0];
	int nxyz = lxy*lz1[0];
	int m0 = lx1[0]-1;
	int nnel = nxyz*nelt[0];
	int lxyzlelt = nxyz*lelt[0];
	int toteqlxyz= toteq[0]*nxyz;
	int toteqlxyzlelt=toteqlxyz*lelt[0];
	int lxz2ldim=lxz*2*ldim[0];
	int ntot=lxz2ldim*lelt[0];
	int lxz2ldimlelt=ntot;


	double *d_ur;
	double *d_us;
	double *d_ut;
	double *d_ud;
	double *d_tmp;

	double *d_hface;
	double *d_normal;

	hipMalloc((void**)&d_ur,nnel*sizeof(double));
	hipMalloc((void**)&d_us,nnel*sizeof(double));
	hipMalloc((void**)&d_ut,nnel*sizeof(double));
	hipMalloc((void**)&d_ud,nnel*sizeof(double));
	hipMalloc((void**)&d_tmp,nnel*sizeof(double));
	hipMalloc((void**)&d_hface,lxz2ldimlelt*sizeof(double));
	hipMalloc((void**)&d_normal,lxz2ldimlelt*sizeof(double));



	for(int eq=0; eq<toteq[0];eq++){

		hipMemset(d_ud, 0.0, nnel*sizeof(double));

		int blockSize1 = glbblockSize1[0], blockSize2=glbblockSize2[0], gridSize1,gridSize2;
		gridSize1 = (int)ceil((float)nnel/blockSize1);
		viscous_cmt_gpu_kernel1<<<gridSize1, blockSize1>>>(d_diffh,d_gradu, d_vtrans,d_vdiff,d_vx,d_vy,d_vz,d_u,d_viscscr,lx1[0],ly1[0],lz1[0], lxy,nxyz,toteq[0], nnel,lxyzlelt,toteqlxyz, toteqlxyzlelt, irho[0], ilam[0],imu[0],icv[0], iknd[0],inus[0],eq,if3d[0],ldim[0] );


		gridSize2= (int)ceil((float)ntot/blockSize2);

		viscous_cmt_gpu_kernel2<<<gridSize2, blockSize2>>>(d_graduf,d_normal,d_unx,d_uny,d_unz,d_iface_flux,d_hface,d_diffh,ntot,lxz2ldim,lxz2ldimlelt,nxyz,eq,lxyzlelt,d_area);

		viscous_cmt_gpu_kernel3<<<gridSize1, blockSize1>>>(d_ur,d_us,d_ut, d_jacmi,d_rxm1,d_rym1,d_rzm1,d_sxm1,d_sym1,d_szm1,d_txm1,d_tym1,d_tzm1,d_diffh,ldim[0],nnel,lxyzlelt,d_bm1,if3d[0]);

		if(if3d){
			gpu_local_grad3_t(d_ud, d_ur, d_us,d_ut,lx1[0],d_dxm1,d_dxtm1, d_tmp, nelt[0]);		
		}
		else{
			gpu_local_grad2_t(d_ud, d_ur, d_us,lx1[0],d_dxm1,d_dxtm1, d_tmp, nelt[0]);		

		}
		viscous_cmt_gpu_kernel4<<<gridSize1, blockSize1>>>(d_res1,d_ud,nnel, eq,lxyzlelt);


	}
	hipFree(d_ur);
	hipFree(d_us);
	hipFree(d_ut);
	hipFree(d_ud);
	hipFree(d_tmp);
	hipFree(d_hface);
	hipFree(d_normal);

	hipError_t code2 = hipPeekAtLastError();

	printf("CUDA: End viscous_cmt_wrapper cuda status: %s\n",hipGetErrorString(code2));
}


__global__ void compute_forcing_gpu_kernel1(double *ur,double *us,double *ut,double *rm1,double *sm1, double *tm1,double *rdumz, double *jacmi,int nnel){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nnel){

		rdumz[id]= 1.0/jacmi[id]*(ur[id]*rm1[id]+us[id]*sm1[id]+ut[id]*tm1[id]);

	}
}

__global__ void compute_forcing_gpu_kernel2(double *ur,double *us,double *rm1,double *sm1,double *rdumz, double *jacm1,int nnel){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nnel){

		rdumz[id]= 1.0/jacm1[id]*(ur[id]*rm1[id]+us[id]*sm1[id]);

	}
}
__global__ void compute_forcing_gpu_kernel3(double *res1,double *usrf,double *rdumz,double *bm1, int nnel, int lx1,int ly1,int lz1, int lxy, int nxyz, int lxyzlelt,int eq){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nnel){
		int ix = id % lx1;
		int iy = (id/lx1)%ly1;
		int iz = (id / (lxy))%lz1;
		int e = id/nxyz;
		int ixyz=id%nxyz;

		res1[eq*lxyzlelt+id]=res1[eq*lxyzlelt+id]-rdumz[id]*bm1[id];
		res1[eq*lxyzlelt+id]=res1[eq*lxyzlelt+id]-usrf[eq*nxyz+ixyz]*bm1[id];

	}
}
__global__ void compute_forcing_gpu_kernel4(double *res1,double *usrf,double *rdumz,double *bm1, int nnel, int lx1,int ly1,int lz1, int lxy, int nxyz, int lxyzlelt, int eq){
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nnel){
		int ix = id % lx1;
		int iy = (id/lx1)%ly1;
		int iz = (id / (lx1*ly1))%lz1;
		int e = id/nxyz;
		int ixyz=id%nxyz;

		res1[eq*lxyzlelt+id]=res1[eq*lxyzlelt+id]-usrf[eq*nxyz+ixyz]*bm1[id];

	}
}

extern "C" void compute_forcing_gpu_wrapper_(int *glbblockSize1,double *d_phig,double *d_rxm1,double *d_sxm1,double *d_txm1,double *d_rym1,double *d_sym1,double *d_tym1,double *d_rzm1,double *d_szm1,double *d_tzm1,double *d_jacmi,double *d_pr,double *d_res1,double *d_usrf,double *d_bm1,int *lx1,int *ly1,int *lz1,int *lelt,int *nelt,int *if3d,int *lxd,int *lyd,int *lzd,int *toteq,int *ldim,double *d_wkd){
	hipDeviceSynchronize();
	hipError_t code1 = hipPeekAtLastError();
	printf("CUDA: Start compute_forcing_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));

	printf("CUDA: Start compute_forcing_gpu_wrapper values, lx1=%d,ly1=%d,lz1=%d,lelt=%d,nelt=%d,if3d=%d,lxd=%d,lyd=%d,lzd=%d,toteq=%d,ldim=%d \n", lx1[0],ly1[0],lz1[0],lelt[0],nelt[0],if3d[0],lxd[0],lyd[0],lzd[0],toteq[0],ldim[0]);


	int nnel = lx1[0]*ly1[0]*lz1[0]*nelt[0];

	int lxy  = lx1[0]*ly1[0];
	int nxyz= lx1[0]*ly1[0]*lz1[0];
	int m0 = lx1[0]-1;
	int ldd= lxd[0]*lyd[0]*lzd[0];
	int lxyzlelt= nxyz*lelt[0];

	double *d_ur;
	double *d_us;
	double *d_ut;
	double *d_rdumz;
	double *d_d;
	double *d_dt;

	int *d_pdg;

	hipMalloc((void**)&d_pdg,nelt[0]*2*lxd[0]*sizeof(int));


	hipMalloc((void**)&d_ur,nnel*sizeof(double));
	hipMalloc((void**)&d_us,nnel*sizeof(double));
	hipMalloc((void**)&d_ut,nnel*sizeof(double));
	hipMalloc((void**)&d_rdumz,nnel*sizeof(double));
	hipMalloc((void**)&d_d,ldd*sizeof(double));  // no lelt here. check  .  adeesha
	hipMalloc((void**)&d_dt,ldd*sizeof(double)); // same as above


	int blockSize = glbblockSize1[0], gridSize;
	gridSize = (int)ceil((float)nnel/blockSize);


	for(int eq=0; eq<toteq[0];eq++){
		int ip=0;
		if(eq!=0&&eq!=4){
			hipDeviceSynchronize();
			code1 = hipPeekAtLastError();
			printf("CUDA: before gpu_gradl_rst compute_forcing_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));

			gpu_gradl_rst(d_ur,d_us,d_ut,d_phig, d_d, d_dt,lxd[0], nelt[0], if3d[0],&ip,d_wkd,d_pdg,lx1[0],lxd[0]);
			hipDeviceSynchronize();
			code1 = hipPeekAtLastError();
			printf("CUDA: after gpu_gradl_rst compute_forcing_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));
			if(if3d){
				gridSize = (int)ceil((float)nnel/blockSize);
				if(eq==1){
					compute_forcing_gpu_kernel1<<<gridSize, blockSize>>>(d_ur,d_us,d_ut,d_rxm1,d_sxm1,d_txm1,d_rdumz,d_jacmi,nnel);
					 hipDeviceSynchronize();
                        code1 = hipPeekAtLastError();
                        printf("CUDA: eq=1 after forcing_gpu_kernel1 compute_forcing_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));

				}
				else if(eq==2){
					compute_forcing_gpu_kernel1<<<gridSize, blockSize>>>(d_ur,d_us,d_ut,d_rym1,d_sym1,d_tym1,d_rdumz,d_jacmi,nnel);
					 hipDeviceSynchronize();
                        code1 = hipPeekAtLastError();
                        printf("CUDA: eq=2 after forcing_gpu_kernel1 compute_forcing_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));

				}
				else if(eq==3){
					compute_forcing_gpu_kernel1<<<gridSize, blockSize>>>(d_ur,d_us,d_ut,d_rzm1,d_szm1,d_tzm1,d_rdumz,d_jacmi,nnel);
					 hipDeviceSynchronize();
                        code1 = hipPeekAtLastError();
                        printf("CUDA: eq=3 after forcing_gpu_kernel1 compute_forcing_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));

				}
			}
			else{
				gridSize = (int)ceil((float)nnel/blockSize);
				if(eq==1){
					compute_forcing_gpu_kernel2<<<gridSize, blockSize>>>(d_ur,d_us,d_rxm1,d_sxm1,d_rdumz,d_jacmi,nnel);

				}
				else if(eq==2){
					compute_forcing_gpu_kernel2<<<gridSize, blockSize>>>(d_ur,d_us,d_rym1,d_sym1,d_rdumz,d_jacmi,nnel);

				}

			}

			hipDeviceSynchronize();
			code1 = hipPeekAtLastError();
			printf("CUDA: after first if compute_forcing_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));
			gpu_nekcol2(glbblockSize1[0],d_rdumz,d_pr,nnel);

			if(eq!=3 || ldim[0]!=2){
				gridSize = (int)ceil((float)nnel/blockSize);
				compute_forcing_gpu_kernel3<<<gridSize, blockSize>>>(d_res1, d_usrf,d_rdumz,d_bm1,nnel,lx1[0],ly1[0],lz1[0],lxy,nxyz,lxyzlelt,eq);

			}
			
		hipDeviceSynchronize();
                	       code1 = hipPeekAtLastError();
                        printf("CUDA: eq<4 end if compute_forcing_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));
		}
		else if (eq==4){
			gridSize = (int)ceil((float)nnel/blockSize);
			compute_forcing_gpu_kernel4<<<gridSize, blockSize>>>(d_res1, d_usrf,d_rdumz,d_bm1,nnel,lx1[0],ly1[0],lz1[0],lxy,nxyz,lxyzlelt,eq);

		}

			hipDeviceSynchronize();
			code1 = hipPeekAtLastError();
			printf("CUDA: eq= %d gpu_gradl_rst ccmpute_forcing_gpu_wrapper cuda status: %s\n",eq,hipGetErrorString(code1));

	}
	hipFree(d_ur);
	hipFree(d_us);
	hipFree(d_ut);
	hipFree(d_d);
	hipFree(d_dt);

	hipFree(d_pdg);

	hipError_t code2 = hipPeekAtLastError();

	printf("CUDA: End compute_forcing_wrapper cuda status: %s\n",hipGetErrorString(code2));

}               


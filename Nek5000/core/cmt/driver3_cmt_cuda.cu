
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#define DEBUGPRINT 0

__global__ void compute_primitive_vars_kernel (double *vx, double *vy, double *vz, double *u, int nelt, int nxyz,int ntot,int irpu, int irpv, int irpw, int iret, int irg, int toteq,int if3d,double *scr, double* energy, double *vtrans, int irho, double *phig, int lx1, int ly1, int lz1, int *lglel, double *xm1, double *ym1, double *zm1, double *t,int ldimt, int npscal, double *pr, double p0th, double *sii, double *siii, double *vdiff, int ifield,char *cb, int icv, int icp, double *csound, int imu,int ilam, double cpgref, double cvgref, double gmaref, double rgasref,  int ltot,int lxy){    
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<nelt*nxyz){

		int e = id/nxyz;
		int i = id%nxyz;
		int e_offset = toteq*nxyz;
		double c = u[e*e_offset+(irg-1)*nxyz+i];
		vx[id] = u[e*e_offset+(irpu-1)*nxyz+i]/c;//invcol3
		vy[id] = u[e*e_offset+(irpv-1)*nxyz+i]/c;
		vz[id] = u[e*e_offset+(irpw-1)*nxyz+i]/c;
//               if(e<5){
//                printf("u is %.20lf  %.20lf %.20lf %.20lf %.20lf e %d i %d  id %d \n",u[e*e_offset+(iret-1)*nxyz+i], u[e*e_offset+(irpu-1)*nxyz+i],u[e*e_offset+(irpv-1)*nxyz+i], u[e*e_offset+(irpw-1)*nxyz+i],u[e*e_offset+(irg-1)*nxyz+i],e,i,id);
//		}

		if(if3d){
			//Compute a Cartesian vector dot product. 3-d version  vdot3
			scr[id] = u[e*e_offset+(irpu-1)*nxyz+i]*u[e*e_offset+(irpu-1)*nxyz+i]+u[e*e_offset+(irpv-1)*nxyz+i]*u[e*e_offset+(irpv-1)*nxyz+i]+u[e*e_offset+(irpw-1)*nxyz+i]*u[e*e_offset+(irpw-1)*nxyz+i];

		}
		else{
			// compute vector dot product 2d version vdot2
			scr[id] = u[e*e_offset+(irpu-1)*nxyz+i]*u[e*e_offset+(irpu-1)*nxyz+i]+u[e*e_offset+(irpv-1)*nxyz+i]*u[e*e_offset+(irpv-1)*nxyz+i];


		}

  //              if(isnan(c)||isnan(-1*c)){
//		  printf("c is nan %.lf e %d i %d  id %d e_offset %d \n",c,e,i,id,e_offset);
//		}

		scr[id] = scr[id]/c; //invcol2
		scr[id] = scr[id] * 0.5; //cmult

		energy[id] =  u[e*e_offset+(iret-1)*nxyz+i] -scr[id];// sub3	
		energy[id] = energy[id]/c;// invcol2
//		if(isnan(energy[id])) {
//		  printf("energy nan %.lf e %d i %d  id %d c %.30lf u %.30lf scr%.30lf \n",energy[id],e,i,id,c,u[e*e_offset+(iret-1)*nxyz+i],scr[id]);
//		}
		vtrans[(irho-1)*ltot+id ] = c / phig[id];  //invcol3

		// subroutine tdstate

		int eg= lglel[e]; // this never uses.  Check with Dr.Tania
		int k =  (id / (lx1*ly1))%lz1;
		int j =  (id/lx1)%ly1;
		int newi = id % lx1;

		double x = xm1[e*nxyz+k*lxy+j*lx1+newi];
		double y = ym1[e*nxyz+k*lxy+j*lx1+newi];
		double z = zm1[e*nxyz+k*lxy+j*lx1+newi];
		double r = x*x+y*y;
		double theta=0.0;
		if (r>0.0){ r = sqrtf(r);}
		if ( x != 0.0 || y!= 0.0){theta = atan2(y,x);	}
		double ux= vx[e*nxyz+k*lxy+j*lx1+newi];
		double uy= vy[e*nxyz+k*lxy+j*lx1+newi];
		double uz= vz[e*nxyz+k*lxy+j*lx1+newi];
		double temp = t [ e*nxyz+k*lxy+j*lx1+newi ];
		int ips;
		double ps[10]; // ps is size of ldimt which is 3. Not sure npscal is also 3. Need to check with Dr.Tania
		for (ips=0;ips<npscal;ips++){
			ps[ips]=t[(ips+1)*ltot+e*nxyz+k*lxy+j*lx1+newi ]; // 5 th dimension of t is idlmt which is 3. Not sure how the  nekasgn access ips+1. Need to check with Dr.Tania
		}
		double pa = pr [e*nxyz+k*lxy+j*lx1+newi];
		double p0= p0th;
		double si2 =  sii[e*nxyz+k*lxy+j*lx1+newi];
		double si3 =  siii[e*nxyz+k*lxy+j*lx1+newi];
		double udiff =  vdiff[(ifield-1)*ltot+e*nxyz+k*lxy+j*lx1+newi];
		double utrans =  vtrans[(ifield-1)*ltot+e*nxyz+k*lxy+j*lx1+newi];
		char cbu1 = cb[0];
		char cbu2 = cb[1];
		char cbu3 = cb[2];

		// subroutine cmtasgn
		int eqnum;
		double varsic[10];
		for (eqnum=0;eqnum<toteq;eqnum++){
			varsic[eqnum] = u[e*e_offset+eqnum*nxyz+k*lxy+j*lx1+newi];

		}
		double phi = phig[e*nxyz+k*lxy+j*lx1+newi];
		double rho = vtrans[(irho-1)*ltot +e*nxyz+k*lxy+j*lx1+newi];
		double pres = pr[e*nxyz+k*lxy+j*lx1+newi];
		double cv=0.0,cp=0.0;
		if(rho!=0){
			cv=vtrans[(icv-1)*ltot +e*nxyz+k*lxy+j*lx1+newi]/rho;
			cp=vtrans[(icp-1)*ltot +e*nxyz+k*lxy+j*lx1+newi]/rho;
		}
		double asnd = csound [e*nxyz+k*lxy+j*lx1+newi];
		double mu = vdiff[(imu-1)*ltot+e*nxyz+k*lxy+j*lx1+newi];
		udiff = vdiff[(imu-1)*ltot+e*nxyz+k*lxy+j*lx1+newi];// this overrides the udiff in nekasgn (line 63 in this function). Need to check withDr.Tania
		double lambda = vdiff[(ilam-1)*ltot+e*nxyz+k*lxy+j*lx1+newi];

		double e_internal = energy[e*nxyz+k*lxy+j*lx1+newi];
		//subroutine cmt_userEOS
		cp=cpgref;
		cv=cvgref;
		temp=e_internal/cv; // overrides
		// function MixtPerf_C_GRT
		asnd=sqrt(gmaref*rgasref*temp);  //overrides
		// function MixtPerf_P_DRT
//		if(isnan(asnd)) {
  //                printf("asnd nan %.30lf e_internal %.30lf e %d i %d  id %d j %d  k %d  newi %d cv %.30lf temp %.30lf gmaref %.30lf rgasref %.30lf sqrt %.30lf \n",asnd,e_internal,e,i,id,j,k,newi,cv,temp,gmaref,rgasref,gmaref*rgasref*temp);
    //            }

		pres=rho*rgasref*temp;//overrides

		vtrans[(icp-1)*ltot +e*nxyz+k*lxy+j*lx1+newi]=cp*rho;
		vtrans[(icv-1)*ltot +e*nxyz+k*lxy+j*lx1+newi]=cv*rho;
		t [ e*nxyz+k*lxy+j*lx1+newi ]= temp;
		csound [e*nxyz+k*lxy+j*lx1+newi]=asnd;

	}
}

extern "C" void compute_primitive_vars_gpu_wrapper_(int *glbblockSize1,double *d_vx, double *d_vy, double *d_vz, double *d_u, int *nxyz, int *ntot, int *nelt,int *irpu, int *irpv, int *irpw, int* iret,  int *irg, int *toteq, int *if3d, double *d_vtrans, int *irho, double *d_phig, int *lx1, int *ly1, int *lz1, int *d_lglel, double *d_xm1, double *d_ym1, double *d_zm1, double *d_t,int *ldimt, int *npscal, double *d_pr, double *p0th, double *d_sii, double *d_siii, double *d_vdiff, int *ifield,char *d_cb, int *icv, int *icp, double *d_csound, int *imu,int *ilam, double *cpgref, double *cvgref, double *gmaref, double *rgasref, int *ltot){
#ifdef DEBUGPRINT
	hipError_t code1 = hipPeekAtLastError();

	printf("CUDA: Start compute_primitive_vars_gpu_wrapper cuda status: %s\n",hipGetErrorString(code1));

	printf("CUDA: Start compute_primitive_vars_gpu_wrapper values nxyz = %d,ntot = %d,nelt = %d,irpu = %d,irpv = %d,irpw = %d,iret = %d,irg=%d ,toteq = %d,if3d = %d,irho = %d,lx1 = %d,ly1 = %d,lz1 = %d,ldimt = %d,npscal = %d,p0th = %lf,ifield = %d,icv = %d,icp = %d,imu = %d,ilam = %d,cpgref = %lf,cvgref = %.20lf,gmaref = %lf,rgasref = %lf,ltot = %d,  \n", nxyz[0],ntot[0],nelt[0],irpu[0],irpv[0],irpw[0],iret[0],irg[0],toteq[0],if3d[0],irho[0],lx1[0],ly1[0],lz1[0],ldimt[0],npscal[0],p0th[0],ifield[0],icv[0],icp[0],imu[0],ilam[0],cpgref[0],cvgref[0],gmaref[0],rgasref[0],ltot[0]);
#endif

	double *d_scr;  // I think this is a tempory variable. need to check with Dr.Tania. adeesha
	double *d_energy;
	hipMalloc((void**)&d_scr,ntot[0] *  sizeof(double));
	hipMalloc((void**)&d_energy,ntot[0] *  sizeof(double));

	int lxy=lx1[0]*ly1[0];
	int blockSize =glbblockSize1[0], gridSize;
	gridSize = (int)ceil((float)nelt[0]*nxyz[0]/blockSize);
	compute_primitive_vars_kernel<<<gridSize, blockSize>>>(d_vx, d_vy, d_vz, d_u, nelt[0], nxyz[0],ntot[0],  irpu[0], irpv[0], irpw[0], iret[0],irg[0],toteq[0],if3d[0],d_scr,d_energy,d_vtrans, irho[0],d_phig ,lx1[0], ly1[0],lz1[0], d_lglel, d_xm1, d_ym1,d_zm1, d_t,ldimt[0], npscal[0], d_pr,p0th[0], d_sii,d_siii,d_vdiff, ifield[0],d_cb, icv[0], icp[0],d_csound,imu[0],ilam[0],  cpgref[0], cvgref[0], gmaref[0], rgasref[0],ltot[0],lxy);


	hipFree(d_scr);
	hipFree(d_energy);
#ifdef DEBUGPRINT
	hipError_t code2 = hipPeekAtLastError();

	printf("CUDA: End compute_primitive_vars_gpu_wrapper cuda status: %s\n",hipGetErrorString(code2));
#endif
}

__global__ void update_u_gpu_kernel(double *u, double *bm1, double *tcoef, double *res3, double *res1, int nelt, int lelt, int lx1, int ly1, int lz1, int toteq, int stage, int lxyz, int lxyznelttoteq, int lxyznelt, int lxyzlelt){//added parameter lxyzlelt by Kk 03/18
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	int ix = id % lx1;
	int iy = (id/lx1)%ly1;
	int iz = (id / (lx1*ly1))%lz1;
	int e = (id/lxyz) % nelt;
	int eq = id/lxyznelt;

	if(id<lxyznelttoteq){



	  u[e*lxyz*toteq + eq*lxyz + ix + iy*lx1 + iz*lx1*ly1] = bm1[ix + iy*lx1 + iz*lx1*ly1 + e*lxyz] * tcoef[(stage-1)*3] * res3[e*lxyz*toteq + eq*lxyz + ix + iy*lx1 + iz*lx1*ly1] + bm1[ix + iy*lx1 + iz*lx1*ly1 + e*lxyz] *  tcoef[(stage-1)*3+1] * u[e*lxyz*toteq + eq*lxyz + ix + iy*lx1 + iz*lx1*ly1] - tcoef[(stage-1)*3+2] * res1[eq*lxyzlelt + e*lxyz + ix+iy*lx1+iz*lx1*ly1];

	  u[e*lxyz*toteq + eq*lxyz + ix + iy*lx1 + iz*lx1*ly1] = u[e*lxyz*toteq + eq*lxyz + ix + iy*lx1 + iz*lx1*ly1] / bm1[ix + iy*lx1 + iz*lx1*ly1 + e*lxyz];

          if(eq ==1 && e == 0 && id%lxyz==10){
              printf("debug u here %25.16E %25.16E %25.16E %25.16E %25.16E %25.16E %25.16E %25.16E %25.16E %25.16E %d %d %d %d %d %d \n", bm1[ix + iy*lx1 + iz*lx1*ly1 + e*lxyz], tcoef[(stage-1)*3], res3[e*lxyz*toteq + eq*lxyz + ix + iy*lx1 + iz*lx1*ly1], tcoef[(stage-1)*3+1],  u[e*lxyz*toteq + eq*lxyz + ix + iy*lx1 + iz*lx1*ly1], tcoef[(stage-1)*3+2], res1[eq*lxyzlelt + e*lxyz + ix+iy*lx1+iz*lx1*ly1], bm1[ix + iy*lx1 + iz*lx1*ly1 + e*lxyz] * tcoef[(stage-1)*3] * res3[e*lxyz*toteq + eq*lxyz + ix + iy*lx1 + iz*lx1*ly1], bm1[ix + iy*lx1 + iz*lx1*ly1 + e*lxyz] *  tcoef[(stage-1)*3+1] * u[e*lxyz*toteq + eq*lxyz + ix + iy*lx1 + iz*lx1*ly1], tcoef[(stage-1)*3+2] * res1[eq*lxyzlelt + e*lxyz + ix+iy*lx1+iz*lx1*ly1], id, ix, iy, iz, e, eq);

          }
//	  printf("debug update_u u  : %.30lf %d %d %d %d %d %d %d %d %d\n",u[e*lxyz*toteq + eq*lxyz + ix + iy*lx1 + iz*lx1*ly1],id,e,eq,ix,iy,iz,blockDim.x,blockIdx.x,threadIdx.x );


	  }


}
extern "C" void update_u_gpu_wrapper_(int *glbblockSize1, double *d_u, double *d_bm1, double *d_tcoef, double *d_res3, double *d_res1, int *nelt, int *lelt, int *lx1, int *ly1, int *lz1, int *toteq, int *stage){

	printf("stagem %d \n",stage[0]);
	printf("values  %d %d %d %d %d %d %d \n",lx1[0],ly1[0],nelt[0],lelt[0],toteq[0],lz1[0],stage[0]);

	int lxyz = lx1[0]*ly1[0]*lz1[0];
	int lxyznelt = lx1[0]*ly1[0]*lz1[0]*nelt[0];
	int lxyzlelt = lx1[0]*ly1[0]*lz1[0]*lelt[0]; //added by Kk 03/18
	int lxyznelttoteq = lx1[0]*ly1[0]*lz1[0]*nelt[0]*toteq[0];
	int blockSize =glbblockSize1[0], gridSize;
	gridSize = (int)ceil((float)nelt[0]*lxyz*toteq[0]/blockSize);
	printf("gridsize ddd %d %d \n",gridSize,blockSize);

	update_u_gpu_kernel<<<gridSize, blockSize>>>(d_u, d_bm1, d_tcoef, d_res3, d_res1, nelt[0], lelt[0], lx1[0], ly1[0], lz1[0], toteq[0], stage[0], lxyz, lxyznelttoteq, lxyznelt, lxyzlelt);

 hipDeviceSynchronize();
       hipError_t  code1 = hipPeekAtLastError();
        printf("CUDA: update_u_gpu_wrapper: cuda status: %s\n",hipGetErrorString(code1));



#ifdef DEBUGPRINT
	hipError_t code2 = hipPeekAtLastError();

	printf("CUDA: End compute_primitive_vars_gpu_wrapper cuda status: %s\n",hipGetErrorString(code2));
#endif

}
